#include "hip/hip_runtime.h"
/*
 * marching_tetra.cu
 *
 *  Created on: Sep 4, 2012
 *      Author: ollie
 */

#include <sys/time.h>

#include <vtkImageData.h>
#include <vtkRTAnalyticSource.h>

#include <piston/vtk_image3d.h>
#include "piston/image3d_to_tetrahedrons.h"
#include "piston/marching_tetrahedron.h"


//#define SPACE thrust::host_space_tag
#define SPACE thrust::detail::default_device_space_tag

using namespace piston;

int
main()
{
    vtkRTAnalyticSource *src = vtkRTAnalyticSource::New();
    src->SetWholeExtent(-100, 100, -100, 100, -100, 100);
    src->Update();

    vtk_image3d<SPACE> image(src->GetOutput());

    // get max and min of 3D scalars
    float min_iso = *thrust::min_element(image.point_data_begin(), image.point_data_end());
    float max_iso = *thrust::max_element(image.point_data_begin(), image.point_data_end());

    typedef image3d_to_tetrahedrons<vtk_image3d<SPACE> > tetra_source;
    tetra_source tetra(image);

    marching_tetrahedron<tetra_source, tetra_source> isosurface(tetra, tetra, 160.0f);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso; isovalue += ((max_iso-min_iso)/50)) {
	isosurface.set_isovalue(isovalue);
	isosurface();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);

    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "total time: " << seconds << ", fps: " << 50.f/seconds << std::endl;
    return 0;

}
