#include "hip/hip_runtime.h"
/*
 * marching_tetra.cu
 *
 *  Created on: Sep 4, 2012
 *      Author: ollie
 */

#include <sys/time.h>

#include <vtkImageData.h>
#include <vtkRTAnalyticSource.h>

#include <piston/vtk_image3d.h>
#include "piston/image3d_to_tetrahedrons.h"
#include "piston/marching_tetrahedron.h"

using namespace piston;

int
main(int argc, char *argv[])
{
    int grid_size = 128;
    if (argc > 1)
	grid_size = atoi(argv[1])/2;

    vtkRTAnalyticSource *src = vtkRTAnalyticSource::New();
    src->SetWholeExtent(-grid_size, grid_size, -grid_size, grid_size,
                        -grid_size, grid_size);
    src->Update();

    vtk_image3d<> image(src->GetOutput());

    // get max and min of scalars
    float min_iso = *thrust::min_element(image.point_data_begin(),
                                         image.point_data_end());
    float max_iso = *thrust::max_element(image.point_data_begin(),
                                         image.point_data_end());

    typedef image3d_to_tetrahedrons<vtk_image3d<> > tetra_source;
    tetra_source tetra(image);

    marching_tetrahedron<tetra_source> isosurface(tetra, tetra);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso;
	 isovalue += ((max_iso-min_iso)/50.0f)) {
	isosurface.set_isovalue(isovalue);
	isosurface();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);

    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "grid_size: " << grid_size*2
	      << ", total time: " << seconds
	      << ", fps: " << 50.0f/seconds << std::endl;
    return 0;

}
