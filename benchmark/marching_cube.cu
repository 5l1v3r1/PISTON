#include "hip/hip_runtime.h"
/*
 * marching_cube.cu
 *
 *  Created on: Sep 7, 2012
 *      Author: ollie
 */

#include <sys/time.h>

#include <vtkImageData.h>
#include <vtkRTAnalyticSource.h>

#include <piston/vtk_image3d.h>
#include "piston/marching_cube.h"

using namespace piston;

int
main(int argc, char *argv[])
{
    int grid_size = 128;
    if (argc > 1)
	grid_size = atoi(argv[1])/2;

    vtkRTAnalyticSource *src = vtkRTAnalyticSource::New();
    src->SetWholeExtent(-grid_size, grid_size, -grid_size, grid_size,
                        -grid_size, grid_size);
    src->Update();

    vtk_image3d<> image(src->GetOutput());

    // get max and min of 3D scalars
    float min_iso = *thrust::min_element(image.point_data_begin(),
                                         image.point_data_end());
    float max_iso = *thrust::max_element(image.point_data_begin(),
                                         image.point_data_end());

    typedef vtk_image3d<> image_source;
    marching_cube<image_source> isosurface(image, image);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso;
	 isovalue += ((max_iso-min_iso)/50)) {
	isosurface.set_isovalue(isovalue);
	isosurface();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);

    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "grid_size: " << grid_size*2
	      << ", total time: " << seconds
	      << ", fps: " << 50.f/seconds << std::endl;
    return 0;

}



