#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <sys/time.h>
#include <cmath>
#include <piston/util/cayley_field.h>
#include <piston/marching_cube.h>

using namespace piston;

int main(int argc, char **argv)
{
    int grid_size = 128;
    if (argc > 1)
	grid_size = atoi(argv[1]);

    cayley_field<> cayley(grid_size, grid_size, grid_size);

    // get max and min of 3D scalars
    float min_iso = *thrust::min_element(cayley.point_data_begin(),
                                         cayley.point_data_end());
    float max_iso = *thrust::max_element(cayley.point_data_begin(),
                                         cayley.point_data_end());

    // create a isosurface filter with cayley as input
    typedef cayley_field<> image_source;
    marching_cube<image_source> contour(cayley, cayley);

    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    for (float isovalue = min_iso; isovalue < max_iso;
	 isovalue += ((max_iso-min_iso)/50)) {
	contour.set_isovalue(isovalue);
	contour();
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "grid_size: " << grid_size*2
	      << ", total time: " << seconds
	      << ", fps: " << 50.f/seconds << std::endl;
    return 0;
}
