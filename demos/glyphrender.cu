#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "glyphrender.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

#define GLYPH_BUFFER_SIZE 12000000


GlyphRender::GlyphRender()
{
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
}


void GlyphRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void GlyphRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void GlyphRender::display()
{
    if (true)
    {
      if (useInterop)
      {
        for (int i=0; i<4; i++) glyphs->vboResources[i] = vboResources[i];
    	glyphs->minValue = minValue;  glyphs->maxValue = maxValue;
      }

      (*(glyphs))();

      if (!useInterop)
      {
    	normals.assign(glyphs->normals_begin(), glyphs->normals_end());
    	indices.assign(glyphs->indices_begin(), glyphs->indices_end());
    	vertices.assign(glyphs->vertices_begin(), glyphs->vertices_end());
    	colors.assign(thrust::make_transform_iterator(glyphs->scalars_begin(), color_map<float>(minValue, maxValue)),
    	              thrust::make_transform_iterator(glyphs->scalars_end(), color_map<float>(minValue, maxValue)));
      }
    }

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 2.0, 0.01, 100.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    /*gluLookAt(center_pos.x, center_pos.y, cameraZ,
                  center_pos.x, center_pos.y, center_pos.z,
                  camera_up.x, camera_up.y, camera_up.z);*/
    gluLookAt(0, 0, 10, 0, 0, 0, 0, 1, 0);
    glPushMatrix();

    //float3 center = make_float3(center_pos.x, center_pos.y, center_pos.z);

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    /*GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float3 offset = make_float3(matrix[0]*center.x + matrix[1]*center.y + matrix[2]*center.z, matrix[4]*center.x + matrix[5]*center.y + matrix[6]*center.z,
                                matrix[8]*center.x + matrix[9]*center.y + matrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;
    glTranslatef(-offset.x, -offset.y, -offset.z);*/

    glColor4f(0.5, 0.5, 0.5, 1.0);

    if (includeInput)
    {
      glColor4f(0.5, 0.5, 0.5, 1.0);
      glEnableClientState(GL_NORMAL_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_VERTEX_ARRAY);

      glNormalPointer(GL_FLOAT, 0, &inputNormalsHost[0]);
      //glColorPointer(4, GL_FLOAT, 0, &inputColorsHost[0]);
      glVertexPointer(3, GL_FLOAT, 0, &inputVerticesHost[0]);
      glDrawElements(GL_TRIANGLES, 3*inputIndicesHost.size(), GL_UNSIGNED_INT, &inputIndicesHost[0]);
    }

    if (includeGlyphs)
    {
      glEnableClientState(GL_VERTEX_ARRAY);
      glEnableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_NORMAL_ARRAY);

      if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboBuffers[3]);
        glVertexPointer(3, GL_FLOAT, 0, 0);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
        glColorPointer(4, GL_FLOAT, 0, 0);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);
        glDrawElements(GL_TRIANGLES, 3*glyphs->numIndices, GL_UNSIGNED_INT, 0);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);
      }
      else
      {
        glNormalPointer(GL_FLOAT, 0, &normals[0]);
        glColorPointer(4, GL_FLOAT, 0, &colors[0]);
        glVertexPointer(3, GL_FLOAT, 0, &vertices[0]);
        glDrawElements(GL_TRIANGLES, 3*indices.size(), GL_UNSIGNED_INT, &indices[0]);
      }
    }

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    glPopMatrix();
}


void GlyphRender::cleanup()
{
	if (useInterop)
	{
	  printf("Deleting VBO\n");
	  if (vboBuffers[0])
	  {
	    for (int i=0; i<4; i++) hipGraphicsUnregisterResource(vboResources[i]);
	    for (int i=0; i<4; i++)
	    {
	      glBindBuffer(1, vboBuffers[i]);
	      glDeleteBuffers(1, &(vboBuffers[i]));
	      vboBuffers[i] = 0;
	    }
	  }
	}
	else
	{
	  vertices.clear(); normals.clear(); colors.clear();
	}
}


void GlyphRender::initGL(bool aAllowInterop)
{
#ifdef USE_INTEROP
    useInterop = aAllowInterop;
#else
    useInterop = false;
#endif

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, 10.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    glMatrixMode(GL_PROJECTION);
    gluPerspective(cameraFOV, 2.0, 200.0, 4000.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
              center_pos.x, center_pos.y, center_pos.z,
              camera_up.x, camera_up.y, camera_up.z);

    if (useInterop)
    {
      glewInit();
      cudaGLSetGLDevice(0);

      // initialize contour buffer objects
      glGenBuffers(4, vboBuffers);
      for (int i=0; i<3; i++)
      {
        unsigned int buffer_size = (i == 2) ? GLYPH_BUFFER_SIZE*sizeof(float3) : GLYPH_BUFFER_SIZE*sizeof(float4);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
      }
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[3]);
      glBufferData(GL_ARRAY_BUFFER, GLYPH_BUFFER_SIZE*sizeof(uint3), 0, GL_DYNAMIC_DRAW);

      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<4; i++) hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
    }

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    read();
}


void GlyphRender::copyPolyData(vtkPolyData *polyData, thrust::device_vector<float3> &points, thrust::device_vector<float3> &vectors, thrust::device_vector<uint3> &indexes)
{
	vtkPoints* pts = polyData->GetPoints();
	vtkFloatArray* verts = vtkFloatArray::SafeDownCast(pts->GetData());
	vtkFloatArray* norms = vtkFloatArray::SafeDownCast(polyData->GetPointData()->GetNormals());
    float3* vData = (float3*)verts->GetPointer(0);
	float3* nData = (float3*)norms->GetPointer(0);
	points.assign(vData, vData+verts->GetNumberOfTuples());
	vectors.assign(nData, nData+norms->GetNumberOfTuples());

	vtkCellArray* cellArray = polyData->GetPolys();
	vtkIdTypeArray* conn = cellArray->GetData();
	vtkIdType* cData = conn->GetPointer(0);
	for (int i=0; i<3*polyData->GetNumberOfPolys(); i++) cData[i] = cData[(i/3)*4+(i%3)+1];
	thrust::host_vector<uint> indexTemp;
	indexTemp.assign(cData, cData+3*polyData->GetNumberOfPolys());
	uint3* c3Data = (uint3*)(thrust::raw_pointer_cast(&*indexTemp.begin()));
	indexes.assign(c3Data, c3Data+polyData->GetNumberOfPolys());
}


int GlyphRender::read()
{
	sphereSource = vtkSphereSource::New();
	sphereSource->Update();
	spherePoly = vtkPolyData::New();
	spherePoly->ShallowCopy(sphereSource->GetOutput());
	copyPolyData(spherePoly, inputVertices, inputNormals, inputIndices);

    arrowSource = vtkArrowSource::New();
    arrowSource->Update();
    triangleFilter = vtkTriangleFilter::New();
    triangleFilter->SetInputConnection(arrowSource->GetOutputPort());
    triangleFilter->Update();
	arrowPoly = vtkPolyData::New();
	arrowPoly->ShallowCopy(triangleFilter->GetOutput());

	normalGenerator = vtkPolyDataNormals::New();
	normalGenerator->SetInput(arrowPoly);
	normalGenerator->ComputePointNormalsOn();
	normalGenerator->ComputeCellNormalsOff();
	normalGenerator->Update();
	arrowPoly = normalGenerator->GetOutput();

	copyPolyData(arrowPoly, glyphVertices, glyphNormals, glyphIndices);

	inputScalars.resize(inputVertices.size());
	maxValue = -FLT_MAX;  minValue = FLT_MAX;
	for (int i=0; i<inputVertices.size(); i++)
	{
		float3 curVertex = inputVertices[i];
		if (curVertex.x > maxValue) maxValue = curVertex.x;
		if (curVertex.x < minValue) minValue = curVertex.x;
	}
	for (int i=0; i<inputScalars.size(); i++)
	{
		float3 curVertex = inputVertices[i];
		inputScalars[i] = 0.2 + 0.8*((curVertex.x - minValue) / (maxValue - minValue)); //0.2 + 1.6*0.01*(rand() % 100);
	}
	minValue = 0.2;  maxValue = 1.0;

	glyphs = new glyph<thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<float>::iterator,
			           thrust::device_vector<float3>::iterator, thrust::device_vector<float3>::iterator, thrust::device_vector<uint3>::iterator>
                      (inputVertices.begin(), inputNormals.begin(), inputScalars.begin(), glyphVertices.begin(), glyphNormals.begin(), glyphIndices.begin(),
                       inputVertices.size(), glyphVertices.size(), glyphIndices.size());

	glyphs->useInterop = useInterop;
	zoomLevelBase = cameraFOV = 40.0; cameraZ = 2.0; zoomLevelPct = zoomLevelPctDefault = 0.5;
	center_pos = make_float3(0, 0, 0);
	cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

	includeGlyphs = true; includeInput = true;
	inputIndicesHost = inputIndices;
	inputNormalsHost = inputNormals;
	inputVerticesHost = inputVertices;

    return 0;
}
