#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

    Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
    Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation
    	and/or other materials provided with the distribution.
    Neither the name of the Los Alamos National Laboratory nor the names of its contributors may be used to endorse or promote products derived from this
    	software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS
OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>

#include "glyphrender.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

#define GLYPH_BUFFER_SIZE 12000000


GlyphRender::GlyphRender()
{
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
}


void GlyphRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void GlyphRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void GlyphRender::display()
{
    if (true)
    {
      /*if (useInterop)
      {
        for (int i=0; i<3; i++) contours[dataSetIndex]->vboResources[i] = vboResources[i];
    	  contours[dataSetIndex]->minIso = minIso;  contours[dataSetIndex]->maxIso = maxIso;
      }*/

      (*(glyphs))();

      if (!useInterop)
      {
    	vertices.assign(glyphs->vertices_begin(), glyphs->vertices_end());
    	normals.assign(glyphs->normals_begin(), glyphs->normals_end());
    	indices.assign(glyphs->indices_begin(), glyphs->indices_end());
      }
    }

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(30.0, 2.0, 0.01, 100.0);
    //gluPerspective(cameraFOV, 2.0, 200.0, 4000.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    /*gluLookAt(center_pos.x, center_pos.y, cameraZ,
                  center_pos.x, center_pos.y, center_pos.z,
                  camera_up.x, camera_up.y, camera_up.z);*/
    gluLookAt(0, 0, 10, 0, 0, 0, 0, 1, 0);
    glPushMatrix();

    //float3 center = make_float3(center_pos.x, center_pos.y, center_pos.z);

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    /*GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float3 offset = make_float3(matrix[0]*center.x + matrix[1]*center.y + matrix[2]*center.z, matrix[4]*center.x + matrix[5]*center.y + matrix[6]*center.z,
                                matrix[8]*center.x + matrix[9]*center.y + matrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;
    glTranslatef(-offset.x, -offset.y, -offset.z);*/

    if (includeGlyphs)
    {
      glEnableClientState(GL_VERTEX_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_NORMAL_ARRAY);

      /*if (useInterop)
      {
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
        glVertexPointer(4, GL_FLOAT, 0, 0);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
        glColorPointer(4, GL_FLOAT, 0, 0);

        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
        glNormalPointer(GL_FLOAT, 0, 0);

        glDrawArrays(GL_TRIANGLES, 0, contours[dataSetIndex]->numTotalVertices);
      }
      else*/
      {
        glNormalPointer(GL_FLOAT, 0, &normals[0]);
        //glColorPointer(4, GL_FLOAT, 0, &colors[0]);
        glVertexPointer(3, GL_FLOAT, 0, &vertices[0]);
        glDrawElements(GL_TRIANGLES, indices.size(), GL_UNSIGNED_INT, &indices[0]);
        //glDrawArrays(GL_TRIANGLES, 0, vertices.size());
      }
    }

    if (includeInput)
    {
      glEnableClientState(GL_NORMAL_ARRAY);
      glDisableClientState(GL_COLOR_ARRAY);
      glEnableClientState(GL_VERTEX_ARRAY);

      glNormalPointer(GL_FLOAT, 0, &inputNormalsHost[0]);
      //glColorPointer(3, GL_FLOAT, 0, colorsx);
      glVertexPointer(3, GL_FLOAT, 0, &inputVerticesHost[0]);
      glDrawElements(GL_TRIANGLES, inputIndicesHost.size(), GL_UNSIGNED_INT, &inputIndicesHost[0]);
    }

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    glPopMatrix();
}


void GlyphRender::cleanup()
{
	if (useInterop)
	{
	  printf("Deleting VBO\n");
	  if (vboBuffers[0])
	  {
	    for (int i=0; i<3; i++) hipGraphicsUnregisterResource(vboResources[i]);
	    for (int i=0; i<3; i++)
	    {
	      glBindBuffer(1, vboBuffers[i]);
	      glDeleteBuffers(1, &(vboBuffers[i]));
	      vboBuffers[i] = 0;
	    }
	  }
	}
	else
	{
	  vertices.clear(); normals.clear(); colors.clear();
	}
}


void GlyphRender::initGL(bool aAllowInterop)
{
#ifdef USE_INTEROP
    useInterop = aAllowInterop;
#else
    useInterop = false;
#endif

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    glMatrixMode(GL_PROJECTION);
    gluPerspective(cameraFOV, 2.0, 200.0, 4000.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
              center_pos.x, center_pos.y, center_pos.z,
              camera_up.x, camera_up.y, camera_up.z);

    if (useInterop)
    {
      glewInit();
      cudaGLSetGLDevice(0);

      // initialize contour buffer objects
      glGenBuffers(3, vboBuffers);
      for (int i=0; i<3; i++)
      {
        unsigned int buffer_size = (i == 2) ? GLYPH_BUFFER_SIZE*sizeof(float3) : GLYPH_BUFFER_SIZE*sizeof(float4);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
      }
      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<3; i++) hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
    }

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    read();
}


void GlyphRender::copyPolyData(vtkPolyData *polyData, thrust::device_vector<float> &points, thrust::device_vector<float> &vectors, thrust::device_vector<GLuint> &indices)
{
	vtkPoints* pts = polyData->GetPoints();
	vtkFloatArray* verts = vtkFloatArray::SafeDownCast(pts->GetData());
	vtkFloatArray* norms = vtkFloatArray::SafeDownCast(polyData->GetPointData()->GetNormals());
    float* vData = verts->GetPointer(0);
	float* nData = norms->GetPointer(0);
	points.assign(vData, vData+verts->GetNumberOfTuples()*3);
	vectors.assign(nData, nData+norms->GetNumberOfTuples()*3);

	vtkCellArray* cellArray = polyData->GetPolys();
	vtkIdTypeArray* conn = cellArray->GetData();
	vtkIdType* cData = conn->GetPointer(0);
	for (int i=0; i<3*polyData->GetNumberOfPolys(); i++) cData[i] = cData[(i/3)*4+(i%3)+1];
	indices.assign(cData, cData+3*polyData->GetNumberOfPolys());
}


int GlyphRender::read()
{
	sphereSource = vtkSphereSource::New();
	sphereSource->Update();
	spherePoly = vtkPolyData::New();
	spherePoly->ShallowCopy(sphereSource->GetOutput());
	copyPolyData(spherePoly, inputVertices, inputNormals, inputIndices);

    arrowSource = vtkArrowSource::New();
    arrowSource->Update();
    triangleFilter = vtkTriangleFilter::New();
    triangleFilter->SetInputConnection(arrowSource->GetOutputPort());
    triangleFilter->Update();
	arrowPoly = vtkPolyData::New();
	arrowPoly->ShallowCopy(triangleFilter->GetOutput());

	normalGenerator = vtkPolyDataNormals::New();
	normalGenerator->SetInput(arrowPoly);
	normalGenerator->ComputePointNormalsOn();
	normalGenerator->ComputeCellNormalsOff();
	normalGenerator->Update();
	arrowPoly = normalGenerator->GetOutput();

	copyPolyData(arrowPoly, glyphVertices, glyphNormals, glyphIndices);

	glyphs = new glyph<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator,
			           thrust::device_vector<float>::iterator, thrust::device_vector<GLuint>::iterator>
                      (inputVertices.begin(), inputNormals.begin(), glyphVertices.begin(), glyphNormals.begin(), glyphIndices.begin(),
                       inputVertices.size(), glyphVertices.size(), glyphIndices.size());

	zoomLevelBase = cameraFOV = 40.0; cameraZ = 2.0; zoomLevelPct = zoomLevelPctDefault = 0.5;
	center_pos = make_float3(0, 0, 0);
	cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

	includeGlyphs = true; includeInput = true;
	inputVerticesHost = inputVertices;
	inputIndicesHost = inputIndices;
	inputNormalsHost = inputNormals;

    return 0;
}
