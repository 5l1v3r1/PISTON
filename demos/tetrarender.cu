#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "tetrarender.h"

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

#define TETRA_BUFFER_SIZE 12000000


TetraRender::TetraRender()
{
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
}


void TetraRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void TetraRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void TetraRender::display()
{
    if (true)
    {
#ifdef USE_INTEROP

      if (useInterop)
      {
        for (int i=0; i<4; i++) isosurface->vboResources[i] = vboResources[i];
        isosurface->minIso = minValue;  isosurface->maxIso = maxValue;
      }
#endif

      ((*isosurface)());

      if (!useInterop)
      {
    	normals.assign(isosurface->normals_begin(), isosurface->normals_end());
    	vertices.assign(isosurface->vertices_begin(), isosurface->vertices_end());
    	colors.assign(thrust::make_transform_iterator(isosurface->scalars_begin(), color_map<float>(minValue, maxValue)),
    	              thrust::make_transform_iterator(isosurface->scalars_end(), color_map<float>(minValue, maxValue)));
      }
    }

    //for (unsigned int i=0; i<vertices.size(); i++) std::cout << vertices[i].x << " " << vertices[i].y << " " << vertices[i].z << " " << std::endl;
    //scalars.assign(isosurface->scalars_begin(), isosurface->scalars_end());
    //for (unsigned int i=0; i<scalars.size(); i++) std::cout << scalars[i] << std::endl;

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT, GL_FILL);
    glPolygonMode(GL_BACK, GL_LINE);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 2.0, 0.01, 100.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(centerPos.x, -10, centerPos.z, centerPos.x, centerPos.y, centerPos.z, 0, 0, 1);
    glPushMatrix();

    qrot.getRotMat(rotationMatrix);
    float3 offset = matrixMul(rotationMatrix, centerPos);

    glMultMatrixf(rotationMatrix);
    glTranslatef(offset.x-centerPos.x, offset.y-centerPos.y, offset.z-centerPos.z);

    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);

    if (useInterop)
    {
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[0]);
      glVertexPointer(4, GL_FLOAT, 0, 0);
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[1]);
      glColorPointer(4, GL_FLOAT, 0, 0);
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[2]);
      glNormalPointer(GL_FLOAT, 0, 0);
      glDrawArrays(GL_TRIANGLES, 0, isosurface->num_total_vertices);
      glBindBuffer(GL_ARRAY_BUFFER, 0);
    }
    else
    {
      if (showIso)
      {
        glNormalPointer(GL_FLOAT, 0, &normals[0]);
        glColorPointer(4, GL_FLOAT, 0, &colors[0]);
        glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
        glDrawArrays(GL_TRIANGLES, 0, vertices.size());
      }
    }

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);

    if ((!useInterop) && (showTets))
    {
      glBegin(GL_TRIANGLES);
      for (unsigned int i=0; i<6; i++)
      {
	if (showTet[i])
	{
	  for (unsigned int j=0; j<12; j++)
	  {
	    glNormal3f(tetNormals[12*i+j].x, tetNormals[12*i+j].y, tetNormals[12*i+j].z);
	    glColor4f(tetColors[12*i+j].x, tetColors[12*i+j].y, tetColors[12*i+j].z, 1.0f);
	    glVertex3f(tetVertices[12*i+j].x, tetVertices[12*i+j].y, tetVertices[12*i+j].z);
	  }
	}
      }
      glEnd();
    }

    glPopMatrix();
}


void TetraRender::cleanup()
{
    if (useInterop)
    {
      printf("Deleting VBO\n");
      if (vboBuffers[0])
      {
        for (int i=0; i<4; i++) hipGraphicsUnregisterResource(vboResources[i]);
	for (int i=0; i<4; i++)
	{
	  glBindBuffer(1, vboBuffers[i]);
	  glDeleteBuffers(1, &(vboBuffers[i]));
	  vboBuffers[i] = 0;
	}
      }
    }
    else
    {
      vertices.clear(); normals.clear(); colors.clear();
    }
}


void TetraRender::initGL(bool aAllowInterop)
{
#ifdef USE_INTEROP
    useInterop = aAllowInterop;
#else
    useInterop = false;
#endif

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, 10.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

#ifdef USE_INTEROP
    if (useInterop)
    {
      glewInit();
      cudaGLSetGLDevice(0);

      // initialize contour buffer objects
      glGenBuffers(4, vboBuffers);
      for (int i=0; i<3; i++)
      {
        unsigned int buffer_size = (i == 2) ? TETRA_BUFFER_SIZE*sizeof(float3) : TETRA_BUFFER_SIZE*sizeof(float4);
        glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[i]);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
      }
      glBindBuffer(GL_ARRAY_BUFFER, vboBuffers[3]);
      glBufferData(GL_ARRAY_BUFFER, TETRA_BUFFER_SIZE*sizeof(uint3), 0, GL_DYNAMIC_DRAW);

      glBindBuffer(GL_ARRAY_BUFFER, 0);
      for (int i=0; i<4; i++) hipGraphicsGLRegisterBuffer(&(vboResources[i]), vboBuffers[i], cudaGraphicsMapFlagsWriteDiscard);
    }
#endif

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    read();
}


void TetraRender::toggleTet(int id)
{
    showTet[id] = !showTet[id];
}


int TetraRender::read()
{
    field = new height_field<int, float, SPACE>(2,2,2);

    thrust::copy(field->point_data_begin(), field->point_data_end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    tetra = new image3d_to_tetrahedrons<height_field<int, float, SPACE> >(*field);

    thrust::copy(tetra->point_data_begin(), tetra->point_data_end(), std::ostream_iterator<int>(std::cout, " "));
    std::cout << std::endl;

    thrust::host_vector<thrust::tuple<int, int, int> > coordinates(tetra->grid_coordinates_begin(), tetra->grid_coordinates_end());
    for (unsigned int i=0; i<coordinates.size(); i++)
	std::cout << thrust::get<0>(coordinates[i]) << " " << thrust::get<1>(coordinates[i]) << " " << thrust::get<2>(coordinates[i]) << " " << std::endl;

    for (unsigned int j=0; j<6; j++)
    {
      float4 curColor;
      if (j == 0) curColor = make_float4(1,0,0,1);
      if (j == 1) curColor = make_float4(0,1,0,1);
      if (j == 2) curColor = make_float4(0,0,1,1);
      if (j == 3) curColor = make_float4(1,1,0,1);
      if (j == 4) curColor = make_float4(0,1,1,1);
      if (j == 5) curColor = make_float4(1,0,1,1);

      float3 v0 = make_float3(thrust::get<0>(coordinates[4*j+0]), thrust::get<1>(coordinates[4*j+0]), thrust::get<2>(coordinates[4*j+0]));
      float3 v1 = make_float3(thrust::get<0>(coordinates[4*j+1]), thrust::get<1>(coordinates[4*j+1]), thrust::get<2>(coordinates[4*j+1]));
      float3 v2 = make_float3(thrust::get<0>(coordinates[4*j+2]), thrust::get<1>(coordinates[4*j+2]), thrust::get<2>(coordinates[4*j+2]));
      float3 v3 = make_float3(thrust::get<0>(coordinates[4*j+3]), thrust::get<1>(coordinates[4*j+3]), thrust::get<2>(coordinates[4*j+3]));

      for (unsigned int i=0; i<12; i++) tetColors.push_back(curColor);

      tetVertices.push_back(make_float4(v0.x, v0.y, v0.z, 1.0f));
      tetVertices.push_back(make_float4(v1.x, v1.y, v1.z, 1.0f));
      tetVertices.push_back(make_float4(v2.x, v2.y, v2.z, 1.0f));
      float3 n0 = cross(v2-v1, v0-v1);
      for (unsigned int i=0; i<3; i++) tetNormals.push_back(n0);

      tetVertices.push_back(make_float4(v2.x, v2.y, v2.z, 1.0f));
      tetVertices.push_back(make_float4(v1.x, v1.y, v1.z, 1.0f));
      tetVertices.push_back(make_float4(v3.x, v3.y, v3.z, 1.0f));
      float3 n1 = cross(v3-v1, v2-v1);
      for (unsigned int i=0; i<3; i++) tetNormals.push_back(n1);

      tetVertices.push_back(make_float4(v3.x, v3.y, v3.z, 1.0f));
      tetVertices.push_back(make_float4(v1.x, v1.y, v1.z, 1.0f));
      tetVertices.push_back(make_float4(v0.x, v0.y, v0.z, 1.0f));
      float3 n2 = cross(v0-v1, v3-v1);
      for (unsigned int i=0; i<3; i++) tetNormals.push_back(n2);

      tetVertices.push_back(make_float4(v0.x, v0.y, v0.z, 1.0f));
      tetVertices.push_back(make_float4(v2.x, v2.y, v2.z, 1.0f));
      tetVertices.push_back(make_float4(v3.x, v3.y, v3.z, 1.0f));
      float3 n3 = cross(v3-v2, v0-v2);
      for (unsigned int i=0; i<3; i++) tetNormals.push_back(n3);

      showTet[j] = true;
    }
    showIso = true;
    showTets = false;
    centerPos = make_float3(0.5, 0.5, 0.5);

    isosurface = new marching_tetrahedron<image3d_to_tetrahedrons<height_field<int, float, SPACE> >,
    			                  image3d_to_tetrahedrons<height_field<int, float, SPACE> > >(*tetra, *tetra, 0.5f);
    isosurface->useInterop = useInterop;
    zoomLevelBase = cameraFOV = 40.0; cameraZ = 2.0; zoomLevelPct = zoomLevelPctDefault = 0.5;
    center_pos = make_float3(0, 0, 0);
    cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

    return 0;
}
