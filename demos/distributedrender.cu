#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/binary_search.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>
#include <sys/time.h>

#include "distributedrender.h"
#include <piston/dthrust.h>


DistributedRender::DistributedRender()
{
}


void DistributedRender::setIsovaluePct(float pct)
{
    isovaluePct = pct;
    isovalue = minIso + pct*(maxIso-minIso);
}



void DistributedRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = zoomLevelBase*pct;
}


void DistributedRender::resetView()
{
    qrot.set(0.0f, 0.0f, 0.0f, 1.0f);
    zoomLevelPct = 0.5f;
    cameraFOV = zoomLevelBase*zoomLevelPct;
}


struct timeval begin, end, diff;
float seconds;
void DistributedRender::display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(cameraFOV, 1.0, 1.0f, 4.0f*fabs(cameraZ-center_pos.z));

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
              center_pos.x, center_pos.y, center_pos.z,
              camera_up.x, camera_up.y, camera_up.z); 
    glPushMatrix();

    float3 center = make_float3(center_pos.x, center_pos.y, center_pos.z);

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    GLfloat matrix[16];
    glGetFloatv(GL_MODELVIEW_MATRIX, matrix);
    float3 offset = make_float3(matrix[0]*center.x + matrix[1]*center.y + matrix[2]*center.z, matrix[4]*center.x + matrix[5]*center.y + matrix[6]*center.z,
                                matrix[8]*center.x + matrix[9]*center.y + matrix[10]*center.z);
    offset.x = center.x - offset.x; offset.y = center.y - offset.y; offset.z = center.z - offset.z;
    glTranslatef(-offset.x, -offset.y, -offset.z);

    glEnableClientState(GL_VERTEX_ARRAY);  
    glDisableClientState(GL_COLOR_ARRAY);   
    glEnableClientState(GL_NORMAL_ARRAY);

    glColor3f(1.0f, 0.0f, 0.0f);     
    glNormalPointer(GL_FLOAT, 0, &normals2[0]);
    //glColorPointer(4, GL_FLOAT, 0, &colors[0]);
    glVertexPointer(4, GL_FLOAT, 0, &vertices2[0]);
    glDrawArrays(GL_TRIANGLES, 0, vertices.size());     

    glPopMatrix();
}


void DistributedRender::cleanup()
{
    vertices.clear(); normals.clear(); colors.clear();
}


void DistributedRender::initContour()
{
    
    int commSize;  MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &commSize));
    int commRank;  MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &commRank));

    if (commRank == 0)
    {
      cayley2 = new tangle_field<SPACE>(GRID_SIZE, GRID_SIZE, GRID_SIZE);
      contour2 = new marching_cube<tangle_field<SPACE>, tangle_field<SPACE> >(*cayley2, *cayley2, 0.46f);

      (*contour2)();
      vertices2.assign(contour2->vertices_begin(), contour2->vertices_end());
      normals2.assign(contour2->normals_begin(), contour2->normals.end());

      thrust::device_vector<int> test1, test2;  test1.resize(24);  test2.resize(12);  for (unsigned int i=0; i<24; i++) test1[i] = i / 3;  
      thrust::upper_bound(test1.begin(), test1.end(), thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(0)+12, test2.begin());
      for (unsigned int i=0; i<test2.size(); i++) std::cout << test2[i] << " ";  std::cout << std::endl;

    }

    cayley = new tangle_field<SPACE>(GRID_SIZE, GRID_SIZE, GRID_SIZE/*, commRank == 0*/);
    contour = new dmarching_cube<tangle_field<SPACE>, tangle_field<SPACE> >(*cayley, *cayley, 0.46f);

    (*contour)();
    
    dthrust::device_to_host(contour->num_total_vertices, contour->vertices, vertices); 
    dthrust::device_to_host(contour->num_total_vertices, contour->normals, normals);  

    int gsize1 = 24;  int gsize2 = 15;
    int lsize1 = gsize1/commSize;  
    thrust::host_vector<int> upinput, upoutput;  thrust::device_vector<int> upinputd, upoutputd;  
    if (commRank == 0) { upinput.resize(gsize1);  for (unsigned int i=0; i<gsize1; i++) upinput[i] = i / 2; }
    dthrust::host_to_device(lsize1, upinput, upinputd);
    dthrust::upper_bound_counting(upinputd.begin(), upinputd.end(), gsize2-1, upoutputd);
    dthrust::device_to_host(upoutputd.size(), upoutputd, upoutput);
    if (commRank == 0) { for (unsigned int i=0; i<upoutput.size(); i++) std::cout << upoutput[i] << " ";  std::cout << std::endl; }


    center_pos = make_float3(0.0f, 0.0f, 0.0f); 
    cameraZ = 5.0f;
    camera_up = make_float3(0.0f, 1.0f, 0.0f);
    zoomLevelBase = 90.0f;
    zoomLevelPct = 0.5f; 
    cameraFOV = zoomLevelPct*zoomLevelBase;
}


void DistributedRender::initGL()
{
    glClearColor(1.0f, 1.0f, 1.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { GRID_SIZE/2.0f, GRID_SIZE/2.0f, 4.0f*GRID_SIZE, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);
}


void DistributedRender::timeContours()
{
    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);
    int numIters = 10;
    for (int i=0; i<numIters; i++)
    {
      isovalue = minIso; // + ((1.0*i)/(1.0*numIters))*(maxIso - minIso);
      //std::cout << "Generating isovalue " << isovalue << std::endl;
      contour->set_isovalue(isovalue);
      
      (*contour)();
    
      //dthrust::device_to_host(contour->num_total_vertices, contour->vertices, vertices); 
      //dthrust::device_to_host(contour->num_total_vertices, contour->normals, normals);
    }
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    std::cout << "contour fps: " << numIters/seconds << std::endl;
}




