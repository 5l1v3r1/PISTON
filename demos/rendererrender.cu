#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <sstream>
#include <float.h>

#include "rendererrender.h"

#define PACKED __attribute__((packed))

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)


struct Rect
{
    int left,top,right,bottom;
};

struct TGAHeader
{
    unsigned char  identsize		;   // size of ID field that follows 18 uint8 header (0 usually)
    unsigned char  colourmaptype	;   // type of colour map 0=none, 1=has palette
    unsigned char  imagetype		;   // type of image 0=none,1=indexed,2=rgb,3=grey,+8=rle packed

    unsigned short colourmapstart	PACKED;   // first colour map entry in palette
    unsigned short colourmaplength	PACKED;   // number of colours in palette
    unsigned char  colourmapbits	;         // number of bits per palette entry 15,16,24,32

    unsigned short xstart		PACKED;   // image x origin
    unsigned short ystart		PACKED;   // image y origin
    unsigned short width		PACKED;   // image width in pixels
    unsigned short height		PACKED;   // image height in pixels
    unsigned char  bits			;         // image bits per pixel 8,16,24,32
    unsigned char  descriptor		;         // image descriptor bits (vh flip bits)

    inline bool IsFlippedHorizontal() const
    {
      return (descriptor & 0x10) != 0;
    }

    inline bool IsFlippedVertical() const
    {
      return (descriptor & 0x20) != 0;
    }
};

#define GLYPH_BUFFER_SIZE 12000000
int rcnt = 0;

RendererRender::RendererRender()
{
    mouse_buttons = 0;
    translate = make_float3(0.0, 0.0, 0.0);
}


void RendererRender::setZoomLevelPct(float pct)
{
    if (pct > 1.0) pct = 1.0;  if (pct < 0.0) pct = 0.0;
    zoomLevelPct = pct;
    cameraFOV = 0.0 + zoomLevelBase*pct;
}


void RendererRender::resetView()
{
    qrot.set(qDefault.x, qDefault.y, qDefault.z, qDefault.w);
    zoomLevelPct = zoomLevelPctDefault;
    cameraFOV = 0.0 + zoomLevelBase*zoomLevelPct;
}


void RendererRender::display()
{
    if (rcnt == 0) (*(renders))();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0*viewportWidth, 0.0, 1.0*viewportHeight, 0.1, 100.0); //gluPerspective(cameraFOV, 2.0, 0.01, 100.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0, 0, 10, 0, 0, 0, 0, 1, 0);
    glPushMatrix();

    //qrot.getRotMat(rotationMatrix);
    //glMultMatrixf(rotationMatrix);

    glColor4f(0.5, 0.5, 0.5, 1.0);

    /*
    glColor4f(0.5, 0.5, 0.5, 1.0);
    glEnableClientState(GL_NORMAL_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glEnableClientState(GL_VERTEX_ARRAY);

    glNormalPointer(GL_FLOAT, 0, &inputNormalsHost[0]);
    //glColorPointer(4, GL_FLOAT, 0, &inputColorsHost[0]);
    glVertexPointer(3, GL_FLOAT, 0, &inputVerticesHost[0]);
    glDrawElements(GL_TRIANGLES, 3*inputIndicesHost.size(), GL_UNSIGNED_INT, &inputIndicesHost[0]);

    glDisableClientState(GL_VERTEX_ARRAY);
    glDisableClientState(GL_COLOR_ARRAY);
    glDisableClientState(GL_NORMAL_ARRAY);*/

    glPopMatrix();

    if (rcnt == 0) screenShot("test.tga", viewportWidth, viewportHeight, true);
    rcnt++;
}


void RendererRender::screenShot(std::string fileName, unsigned int width, unsigned int height, bool includeAlpha)
{
    std::cout << "Screen shot" << std::endl;
    unsigned int pixelSize = 3;
    unsigned int pixelSizeBits = 24;
    GLenum pixelFormat = GL_BGR_EXT;

    if (includeAlpha)
    {
      pixelSize = sizeof(unsigned int);
      pixelSizeBits = 32;
      pixelFormat = GL_BGRA_EXT;
    }

    /*char* pBuffer = new char[pixelSize*width*height ];

    std::cout << "Size: " << pixelSize << std::endl;

    for (unsigned int i=0; i<width; i++)
    {
      for (unsigned int j=0; j<height; j++)
      {
        pBuffer[i*height*pixelSize + j*pixelSize + 0] = 0;
        pBuffer[i*height*pixelSize + j*pixelSize + 1] = 0;
        pBuffer[i*height*pixelSize + j*pixelSize + 2] = 255;
        if (includeAlpha) pBuffer[i*height*pixelSize + j*pixelSize + 3] = 255;
      }
    }*/

    //glReadPixels( 0,0,width,height,pixelFormat,GL_UNSIGNED_BYTE,pBuffer );

    TGAHeader tgah;
    memset( &tgah,0,sizeof(TGAHeader) );

    tgah.bits = pixelSizeBits;
    tgah.height = height;
    tgah.width = width;
    tgah.imagetype = 2;

    std::ofstream ofile( fileName.c_str(), std::ios_base::binary );

    ofile.write( (char*)&tgah, sizeof(tgah) );
    thrust::host_vector<char> hostFrame;
    hostFrame.assign(renders->frame_begin(), renders->frame_end());
    ofile.write( &hostFrame[0], pixelSize*width*height );

    ofile.close();

    //delete [] pBuffer;
}


void RendererRender::cleanup()
{

}


void RendererRender::initGL(bool aAllowInterop)
{
    viewportWidth = 128;  viewportHeight = 128;
    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    float white[] = { 0.5, 0.5, 0.5, 1.0 };
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0, 0.0, 10.0, 1.0 };
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);
    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, black);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    glMatrixMode(GL_PROJECTION);
    gluPerspective(cameraFOV, 2.0, 200.0, 4000.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(center_pos.x, center_pos.y, cameraZ,
              center_pos.x, center_pos.y, center_pos.z,
              camera_up.x, camera_up.y, camera_up.z);

    //printf("Error code: %s\n", hipGetErrorString(errorCode));
    read();
}


int RendererRender::read()
{
    inputVertices.push_back(make_float3(10.0f, 10.0f, 0.0f));
    inputVertices.push_back(make_float3(50.0f, 11.0f, 0.0f));
    inputVertices.push_back(make_float3(11.0f, 50.0f, 0.0f));

    std::cout << "Viewport size: " << viewportWidth << " " << viewportHeight << std::endl;

    renders = new render<thrust::device_vector<float3>::iterator>(inputVertices.begin(), inputVertices.size(), viewportWidth, viewportHeight);

    zoomLevelBase = cameraFOV = 40.0; cameraZ = 2.0; zoomLevelPct = zoomLevelPctDefault = 0.5;
    center_pos = make_float3(0, 0, 0);
    cameraFOV = zoomLevelBase*zoomLevelPct;  camera_up = make_float3(0,1,0);

    inputVerticesHost = inputVertices;

    return 0;
}
