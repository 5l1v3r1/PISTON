#include "hip/hip_runtime.h"
/*
 * my_simple_opengl.cu
 *
 *  Created on: Oct 7, 2011, 2/27/2012
 *      Author: ollie/thorp
 */

#include <iostream>


#ifdef __APPLE__
/* Location of some include headers on the Apple systems:
 * /System/Library/Frameworks/FW.framework/Headers
*/
    #include <GL/glew.h>
    //#include <OpenGL/OpenGL.h>
    #include <OpenGL.framework/Headers/OpenGL.h>
    //#include <OpenGL.h>

    //#include <GLUT/glut.h>
    #include <GLUT.framework/Headers/glut.h>
    //#include <glut.h>
//
//
#else
    #include <GL/glew.h>
    #include <GL/glut.h>
    #include <GL/gl.h>
#endif

#include <cuda_gl_interop.h>

#include <thrust/device_vector.h>
#include <thrust/distance.h>

#include <piston/implicit_function.h>
#include <piston/image2d.h>

//#include <piston/cutil_math.h>
#include <piston/piston_math.h>

//#define SPACE  thrust::host_space_tag
#define SPACE thrust::detail::default_device_space_tag

using namespace piston;
static const int GRID_SIZE = 4;

struct sine_wave: public piston::image2d<int, float4, SPACE>
{
	struct sine_functor : public piston::implicit_function2d<int, float4>
	{
		typedef piston::implicit_function2d<int, float4> Parent;
		typedef typename Parent::InputType InputType;

		int xdim;
		int ydim;
		float time;

		sine_functor(int xdim, int ydim, float time) :
			xdim(xdim), ydim(ydim), time(time) {}

		__host__ __device__
		float4 operator()(InputType pos) const {
			unsigned int x = thrust::get<0>(pos);
			unsigned int y = thrust::get<1>(pos);

			// calculate uv coordinates
			float u = x / (float) xdim;
			float v = y / (float) ydim;
			u = u*2.0f - 1.0f;
			v = v*2.0f - 1.0f;

			// calculate simple sine wave pattern
			float freq = 4.0f;

			float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

			// write output vertex
			return make_float4(u, w, v, 1.0f);
		}
	};

	typedef piston::image2d<int, float4, SPACE> Parent;
	typedef thrust::transform_iterator<sine_functor, typename Parent::GridCoordinatesIterator> PointDataIterator;
	float time;
	PointDataIterator point_data_iterator;

	sine_wave(int xdim, int ydim, float time = 0.0f) :
		Parent(xdim, ydim),
		time(time),
		point_data_iterator(this->grid_coordinates_iterator, sine_functor(xdim, ydim, time)){}

	void resize(int xdim, int ydim) {
		Parent::resize(xdim, ydim);
		point_data_iterator = thrust::make_transform_iterator(grid_coordinates_iterator,
				sine_functor(xdim, ydim, time));
	}

	void set_time(float time) {
		this->time = time;
		point_data_iterator = thrust::make_transform_iterator(grid_coordinates_iterator,
				sine_functor(xdim, ydim, time));
	}

	PointDataIterator point_data_begin() {
		return point_data_iterator;
	}

	PointDataIterator point_data_end() {
		return point_data_iterator + this->NPoints;
	}
};


struct print_tuple2 : public thrust::unary_function<thrust::tuple<int, int>, void>
{
	//__host__ __device__
	void operator() (thrust::tuple<int, int> pos) {
		std::cout << "(" << thrust::get<0>(pos) << ", "
				<< thrust::get<1>(pos) << ")" << std::endl;
	}
};


struct print_float4 : public thrust::unary_function<float4, void>
{
	//__host__ __device__
	void operator() (float4 p) {
	    std::cout << "(" << p.x << ", " << p.y << ", " << p.z << ", " << p.w <<")" << std::endl;
	}
};

/// Extracts a dimension of the space, x dim
struct extractY : public thrust::unary_function<float4, float>
{
	//__host__ __device__
	float operator() (float4 p) {
	    return( p.y);
	}

};

/// Extracts a dimension of the space, x dim
struct extractX : public thrust::unary_function<float4, float>
{
	//__host__ __device__
	float operator() (float4 p) {
	    return( p.x);
	}

};

/// Extracts a dimension of the space, x dim
struct extractZ : public thrust::unary_function<float4, float>
{
	//__host__ __device__
	float operator() (float4 p) {
	    return( p.z);
	}

};

/// Extracts a dimension of the space, x dim
struct extractW : public thrust::unary_function<float4, float>
{
	//__host__ __device__
	float operator() (float4 p) {
	    return( p.w);
	}

};


#if 0
bool init_gl(void)
{
    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, g_window_width, g_window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)g_window_width / (GLfloat) g_window_height, 0.1, 10.0);

    return true;
} // end init_gl

void display(void)
{
    // transform the mesh
    thrust::counting_iterator<int> first(0);
    thrust::counting_iterator<int> last(g_mesh_width * g_mesh_height);

    thrust::transform(first, last,
                      g_vec.begin(),
                      sine_wave(g_mesh_width,g_mesh_height,g_anim));

    // map the vector into GL
    thrust::device_ptr<float4> ptr = &g_vec[0];

    // pass the device_ptr to the allocator's static function map_buffer
    // to map it into GL
    GLuint buffer = gl_vector::allocator_type::map_buffer(ptr);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, g_translate_z);
    glRotatef(g_rotate_x, 1.0, 0.0, 0.0);
    glRotatef(g_rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, g_mesh_width * g_mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();
    glutPostRedisplay();

    g_anim += 0.001;

    // unmap the vector from GL
    gl_vector::allocator_type::unmap_buffer(buffer);
} // end display

void mouse(int button, int state, int x, int y)
{
    if(state == GLUT_DOWN)
    {
	g_mouse_buttons |= 1<<button;
    } // end if
    else if(state == GLUT_UP)
    {
	g_mouse_buttons = 0;
    } // end else if

    g_mouse_old_x = x;
    g_mouse_old_y = y;
    glutPostRedisplay();
} // end mouse

void motion(int x, int y)
{
    float dx, dy;
    dx = x - g_mouse_old_x;
    dy = y - g_mouse_old_y;

    if(g_mouse_buttons & 1)
    {
	g_rotate_x += dy * 0.2;
	g_rotate_y += dx * 0.2;
    } // end if
    else if(g_mouse_buttons & 4)
    {
	g_translate_z += dy * 0.01;
    } // end else if

    g_mouse_old_x = x;
    g_mouse_old_y = y;
} // end motion

void keyboard(unsigned char key, int, int)
{
    switch(key)
    {
    // catch 'esc'
    case(27):
	      // deallocate memory
	      g_vec.clear();
    g_vec.shrink_to_fit();
    exit(0);
    default:
	break;
    } // end switch
} // end keyboard

int main(int argc, char** argv)
{
    // Create GL context
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(g_window_width, g_window_height);
    glutCreateWindow("Thrust/GL interop");

    // initialize GL
    if(!init_gl())
    {
	throw std::runtime_error("Couldn't initialize OpenGL");
    } // end if

    // register callbacks
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);

    // resize the vector to fit the mesh
    g_vec.resize(g_mesh_width * g_mesh_height);

    // transform the mesh
    thrust::counting_iterator<int,thrust::device_space_tag> first(0);
    thrust::counting_iterator<int,thrust::device_space_tag> last(g_mesh_width * g_mesh_height);

    thrust::transform(first, last,
                      g_vec.begin(),
                      sine_wave(g_mesh_width,g_mesh_height,g_anim));

    // start rendering mainloop
    glutMainLoop();

    return 0;
} // end main

#else
int main()
{
	// H has storage for 4 integers
	thrust :: host_vector <int > H (4);

	// initialize individual elements
	H [0] = 14;
	H [1] = 20;
	H [2] = 38;
	H [3] = 46;

	// H. size () returns the size of vector H
	std :: cout << "H has size " << H. size () << std :: endl ;

	// print contents of H
	for ( int i = 0; i < H. size (); i ++)
	std :: cout << "H[" << i << "] = " << H[i] << std :: endl ;

	int maxVal = thrust::reduce(H.begin(), H.end(), 0, thrust::maximum<int>());

	std :: cout << "H has max: " << maxVal << std :: endl ;


	// resize H
	H. resize (2) ;
	std :: cout << "H now has size " << H. size () << std :: endl ;

	// print contents of H again
	for ( int i = 0; i < H. size (); i ++)
	std :: cout << "H[" << i << "] = " << H[i] << std :: endl ;

	// Copy host_vector H to device_vector D
	thrust :: device_vector <int > D = H;

	// elements of D can be modified
	D [0] = 99;
	//D [1] = 88;

	// print contents of D
	for ( int i = 0; i < D. size (); i ++)
		std :: cout << "D[" << i << "] = " << D[i] << std :: endl ;

	/** end of thrust sample code.
	*/

	/**
	 * start of basic piston code.
	 */

	// Declare the basic data space.
	sine_wave field(GRID_SIZE, GRID_SIZE);

	// Declare a host vector to represent the coordinates of all items in the field.
	// The coordinates are (x,y) pairs.
    thrust::host_vector
    <thrust::tuple<int, int> >
    position(field.grid_coordinates_begin(), field.grid_coordinates_end());

    // forall coordinates, from the host, get the data from a specific device location and print it out.
    thrust::for_each(position.begin(), position.end(), print_tuple2());
    /* Also works.
	for ( int i = 0; i < position.size (); i ++)
	std :: cout << "position[" << i << "] = " << "(" << thrust::get<0>(position[i]) << ", "
	                                                      				<< thrust::get<1>(position[i]) << ")"<< std :: endl ;
    */


    // Make a host vector with as many entries as there are in the grid.
    // each entry in the vector has for component values (x, y, z, w)
    // init it with the field point data.
    thrust::host_vector<float4> points(field.point_data_begin(), field.point_data_end());

    std::cout << "test2" << std::endl ;

    // print out the points x,y,z,w values
    thrust::for_each(points.begin(), points.end(), print_float4());




    // Double the size of the grid in both dimentions
    field.resize(2*GRID_SIZE, 2*GRID_SIZE);

    // Resize the position vector to the size of the field.
    /*
     * workspace
     *     typedef typename thrust::counting_iterator<IndexType, MemorySpace> CountingIterator;
     *     typedef typename thrust::transform_iterator<grid_coordinates_functor, CountingIterator> GridCoordinatesIterator;
     *
     *     HAVE:
     *     thrust::transform_iterator<grid_coordinates_functor, thrust::counting_iterator<int, thrust::detail::default_device_space_tag>>
     *
     *     NEED:
     *     thrust::host_vector<int>::iterator
     */
    //thrust::transform_iterator<piston::image2d<int, float4, SPACE>::grid_coordinates_functor, thrust::counting_iterator<int, thrust::detail::default_device_space_tag>> iterBegin= field.grid_coordinates_begin();
    //thrust::transform_iterator<piston::image2d<int, float4, SPACE>::grid_coordinates_functor, thrust::counting_iterator<int, thrust::detail::default_device_space_tag>> iterEnd= field.grid_coordinates_end();
    //  unsigned long int foo= thrust::distance(field.image2d::grid_coordinates_begin(), field.image2d::grid_coordinates_end());
    unsigned long int foo= thrust::distance(field.grid_coordinates_begin(), field.grid_coordinates_end());
    position.resize(foo);
    points.resize(thrust::distance(field.piston::image2d<int,float4,SPACE>::grid_coordinates_begin(), field.image2d<int,float4,SPACE>::grid_coordinates_end()));

    std::cout << "test3" << std::endl ;

    thrust::copy(field.grid_coordinates_begin(), field.grid_coordinates_end(), position.begin());
    thrust::for_each(position.begin(), position.end(), print_tuple2());
    thrust::copy(field.point_data_begin(), field.point_data_end(), points.begin());
    thrust::for_each(points.begin(), points.end(), print_float4());




    std::cout << "Test4: Find the bounding box. " << std::endl ;

    /*
     * Find X dimension bounds.
     */
    thrust::host_vector<float> xVal(foo);
    thrust::transform(points.begin(), points.end(), xVal.begin(), extractX());

	// print contents of D
	for ( int i = 0; i < xVal. size (); i ++)
		std :: cout << "xVal[" << i << "] = " << xVal[i] << std :: endl ;

	//Device vector
	thrust::device_vector<float> xVals(foo);

	// Move to GPU
	xVals=xVal;

    float maxX= thrust::reduce(xVals.begin(), xVals.end(), -1.0f, thrust::maximum<float>());
    float minX= thrust::reduce(xVals.begin(), xVals.end(), +1.0f, thrust::minimum<float>());
    std::cout << "Bounds for X axis = { " << minX << "," << maxX  << "}" << std::endl ;


    /*
     * Find Y dimension bounds.
     */
    thrust::host_vector<float> yVals(foo);
    thrust::transform(points.begin(), points.end(), yVals.begin(), extractY());

    float maxY= thrust::reduce(yVals.begin(), yVals.end(), -1.0f, thrust::maximum<float>());
    float minY= thrust::reduce(yVals.begin(), yVals.end(), +1.0f, thrust::minimum<float>());
    std::cout << "Bounds for Y axis = { " << minY << "," << maxY  << "}" << std::endl ;




    /*
     * Find Z dimension bounds.
     */
    thrust::host_vector<float> zVals(foo);
    thrust::transform(points.begin(), points.end(), zVals.begin(), extractZ());

    float maxZ= thrust::reduce(zVals.begin(), zVals.end(), -1.0f, thrust::maximum<float>());
    float minZ= thrust::reduce(zVals.begin(), zVals.end(), +1.0f, thrust::minimum<float>());
    std::cout << "Bounds for Z axis = { " << minZ << "," << maxZ  << "}" << std::endl ;





    /*
     * Find Y dimension bounds.
     */
    thrust::host_vector<float> wVals(foo);
    thrust::transform(points.begin(), points.end(), wVals.begin(), extractW());

    float maxW= thrust::reduce(wVals.begin(), wVals.end(), -1.0f, thrust::maximum<float>());
    float minW= thrust::reduce(wVals.begin(), wVals.end(), +1.0f, thrust::minimum<float>());
    std::cout << "Bounds for W axis = { " << minW << "," << maxW  << "}" << std::endl ;








    return 0;
} // end main
#endif
