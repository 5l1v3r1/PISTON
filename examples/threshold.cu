#include "hip/hip_runtime.h"
/*
 * threshold.cu
 *
 *  Created on: Sep 21, 2011
 *      Author: ollie
 */

#include <thrust/device_vector.h>
#if 1
#include <GL/glut.h>
#include <piston/sphere.h>
#include <piston/threshold_geometry.h>

//#define SPACE  thrust::host_space_tag
#define SPACE thrust::detail::default_device_space_tag

using namespace piston;
static const int GRID_SIZE = 256;

#if 1
template <typename IndexType, typename ValueType>
struct height_field : public piston::image3d<IndexType, ValueType, SPACE>
{
    struct height_functor : public piston::implicit_function3d<IndexType, ValueType> {
	typedef piston::implicit_function3d<IndexType, ValueType> Parent;
	typedef typename Parent::InputType InputType;

	__host__ __device__
	ValueType operator()(InputType pos) const {
	    return thrust::get<2>(pos);
	};
    };

    typedef piston::image3d<IndexType, ValueType, SPACE> Parent;

    typedef thrust::transform_iterator<height_functor,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    height_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     height_functor()){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter + this->NPoints;
    }
};
#endif

#if 0

template <typename IndexType, typename ValueType>
struct sphere_field : public piston::image3d<IndexType, ValueType, SPACE>
{
    typedef piston::image3d<IndexType, ValueType, SPACE> Parent;

    typedef thrust::transform_iterator<piston::sphere<IndexType, ValueType>,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    sphere_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     piston::sphere<IndexType, ValueType>(xdim/2, ydim/2, zdim/2, 1)){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter+this->NPoints;
    }
};

#else

template <typename IndexType, typename ValueType>
struct sphere_field : public piston::image3d<IndexType, ValueType, SPACE>
{
    typedef piston::image3d<IndexType, ValueType, SPACE> Parent;

//    typedef thrust::host_vector<thrust::tuple<IndexType, IndexType, IndexType> > GridCoordinatesContainer;
    typedef typename choose_container<typename Parent::CountingIterator, thrust::tuple<IndexType, IndexType, IndexType> >::type GridCoordinatesContainer;
    GridCoordinatesContainer grid_coordinates_vector;
    typedef typename GridCoordinatesContainer::iterator GridCoordinatesIterator;
    GridCoordinatesIterator  grid_coordinates_iterator;

//    typedef thrust::host_vector<ValueType> PointDataContainer;
    typedef typename choose_container<typename Parent::CountingIterator, ValueType>::type PointDataContainer;
    PointDataContainer point_data_vector;
    typedef typename PointDataContainer::iterator PointDataIterator;
    PointDataIterator point_data_iterator;

    sphere_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	grid_coordinates_vector(Parent::grid_coordinates_begin(), Parent::grid_coordinates_end()),
	grid_coordinates_iterator(grid_coordinates_vector.begin()),
	point_data_vector(thrust::make_transform_iterator(grid_coordinates_iterator, sphere<IndexType, ValueType>(xdim/2, ydim/2, zdim/2, 1)),
	                  thrust::make_transform_iterator(grid_coordinates_iterator, sphere<IndexType, ValueType>(xdim/2, ydim/2, zdim/2, 1))+this->NPoints),
	point_data_iterator(point_data_vector.begin()) {}

    GridCoordinatesIterator grid_coordinates_begin() {
	return grid_coordinates_iterator;
    }
    GridCoordinatesIterator grid_coordinates_end() {
	return grid_coordinates_iterator+this->NPoints;
    }

    PointDataIterator point_data_begin() {
	return point_data_iterator;
    }
    PointDataIterator point_data_end() {
	return point_data_iterator+this->NPoints;
    }
};

#endif

struct threshold_between : thrust::unary_function<float, bool>
{
    float min_value;
    float max_value;

    threshold_between(float min_value, float max_value) :
	min_value(min_value), max_value(max_value) {}

    __host__ __device__
    bool operator() (float val) const {
	return (min_value <= val) && (val <= max_value);
    }
};

struct print_float4 : public thrust::unary_function<float4, void>
{
	__host__ __device__
	void operator() (float4 p) {
//	    std::cout << "(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
	}
};


int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0, 0, 0.0);
float3 translate = make_float3(0.0, 0.0, 0.0);

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
	mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
	mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
	rotate.x += dy * 0.2;
	rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
	translate.x += dx * 0.01;
	translate.y -= dy * 0.01;
    } else if (mouse_buttons==4) {
	translate.z += dy * 0.1;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

bool wireframe = false;
bool animate = true;
void keyboard( unsigned char key, int x, int y )
{
    switch (key) {
    case 'w':
	wireframe = !wireframe;
	break;
    case 'a':
	animate = !animate;
	break;
    }
}


threshold_geometry<sphere_field<int, float> > *threshold_p;

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float4>
{
    const ValueType min;
    const ValueType max;

    __host__ __device__
    color_map(ValueType min, ValueType max) :
	min(min), max(max) {}

    __host__ __device__
    float4 operator()(ValueType val) {
	// HSV rainbow for height field, stolen form Manta
	const float V = 0.7f, S = 1.0f;
	float H = (1.0f - static_cast<float> (val) / (max - min));

	if (H < 0.0f)
	    H = 0.0f;
	else if (H > 1.0f)
	    H = 1.0f;
	H *= 4.0f;

	float i = floor(H);
	float f = H - i;

	float p = V * (1.0 - S);
	float q = V * (1.0 - S * f);
	float t = V * (1.0 - S * (1 - f));

	float R, G, B;
	if (i == 0.0) {
	    R = V;
	    G = t;
	    B = p;
	} else if (i == 1.0) {
	    R = q;
	    G = V;
	    B = p;
	} else if (i == 2.0) {
	    R = p;
	    G = V;
	    B = t;
	} else if (i == 3.0) {
	    R = p;
	    G = q;
	    B = V;
	} else if (i == 4.0) {
	    R = t;
	    G = p;
	    B = V;
	} else {
	    // i == 5.0
	    R = V;
	    G = p;
	    B = q;
	}
	return make_float4(R, G, B, 1.0);
    }
};

struct tuple2float4 : thrust::unary_function<thrust::tuple<int, int, int>, float4>
{
	__host__ __device__
	float4 operator()(thrust::tuple<int, int, int> xyz) {
	    return make_float4((float) thrust::get<0>(xyz),
	                       (float) thrust::get<1>(xyz),
	                       (float) thrust::get<2>(xyz),
	                       1.0f);
	}
};

void display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) {
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    } else {
	glPolygonMode(GL_BACK, GL_FILL);
	glPolygonMode(GL_FRONT, GL_FILL);
    }

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    glPushMatrix();

    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);
    glTranslatef(-(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2);
    glTranslatef(translate.x, translate.y, translate.z);

    thrust::host_vector<float4> vertices(thrust::make_transform_iterator(threshold_p->vertices_begin(), tuple2float4()),
                                         thrust::make_transform_iterator(threshold_p->vertices_end(),   tuple2float4()));
    thrust::host_vector<float4> colors(thrust::make_transform_iterator(threshold_p->scalars_begin(), color_map<float>(4.0f, 256.0f)),
                                       thrust::make_transform_iterator(threshold_p->scalars_end(),  color_map<float>(4.0f, 256.0f)));

//    glColor4f(1.0f, 1.0f, 1.0f, 1.0f);

//    glNormalPointer(GL_FLOAT, 0, &normals[0]);
    glColorPointer(4, GL_FLOAT, 0, &colors[0]);
    glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
    glDrawArrays(GL_QUADS, 0, vertices.size());

    // set view matrix for 2D message
    // TBD
    glutSwapBuffers();
}

void idle()
{
    if (animate) {
//	isovalue += delta;
//	if (isovalue > maxiso)
//	    delta = -0.05;
//	if (isovalue < miniso)
//	    delta = 0.05;
//	glutPostRedisplay();
    }
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Marching Cube");

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
//    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
//    float ambient[] = { 0.5, 0.0, 0.0, 1.0 };
//    float diffuse[] = { 0.5, 0.0, 0.0, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

//    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, ambient);
//    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse);
//    glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
//    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, ambient);
//    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
//    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    /* Setup the view of the cube. */
    glMatrixMode(GL_PROJECTION);
    gluPerspective( /* field of view in degree */ 60.0,
                    /* aspect ratio */ 1.0,
                    /* Z near */ 1.0, /* Z far */ GRID_SIZE*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, GRID_SIZE*1.5,  /* eye is at (0,0, 1.5*GRID_SIZE) */
              0.0, 0.0, 0.0,		/* center is at (0,0,0) */
              0.0, 1.0, 0.0);		/* up is in positive Y direction */
    glPushMatrix();

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
//    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);

//    glutReshapeFunc( reshape);
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
//    glutIdleFunc(idle);
    glutMainLoop();
}

#endif

int main(int argc, char *argv[])
{

    sphere_field<int, float> scalar_field(GRID_SIZE, GRID_SIZE, GRID_SIZE);
//    thrust::copy(scalar_field.point_data_begin(), scalar_field.point_data_end(), std::ostream_iterator<float>(std::cout, " "));
//    std::cout << std::endl;

    threshold_geometry<sphere_field<int, float> > threshold(scalar_field, 4, 1600);
//    for (int i = 0; i < 10; i++)
	threshold();

//    threshold_p = &threshold;

//    initGL(argc, argv);

    return 0;
}
