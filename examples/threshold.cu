#include "hip/hip_runtime.h"
/*
 * threshold.cu
 *
 *  Created on: Sep 21, 2011
 *      Author: ollie
 */

#include <piston/sphere.h>
#include <piston/threshold_geometry.h>

static const int GRID_SIZE = 5;
template <typename IndexType, typename ValueType>
struct height_field : public piston::image3d<IndexType, ValueType, thrust::host_space_tag>
{
    struct height_functor : public piston::implicit_function3d<IndexType, ValueType> {
	typedef piston::implicit_function3d<IndexType, ValueType> Parent;
	typedef typename Parent::InputType InputType;

	__host__ __device__
	ValueType operator()(InputType pos) const {
	    return thrust::get<2>(pos);
	};
    };

    typedef piston::image3d<IndexType, ValueType, thrust::host_space_tag> Parent;

    typedef thrust::transform_iterator<height_functor,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    height_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     height_functor()){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter + this->NPoints;
    }
};

template <typename IndexType, typename ValueType>
struct sfield : public piston::image3d<IndexType, ValueType, thrust::host_space_tag>
{
    typedef piston::image3d<IndexType, ValueType, thrust::host_space_tag> Parent;

    typedef thrust::transform_iterator<piston::sphere<IndexType, ValueType>,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    sfield(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     piston::sphere<IndexType, ValueType>(0, 0, 0, 1)){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter+this->NPoints;
    }
};

struct threshold_between : thrust::unary_function<float, bool>
{
    float min_value;
    float max_value;

    threshold_between(float min_value, float max_value) :
	min_value(min_value), max_value(max_value) {}

    __host__ __device__
    bool operator() (float val) const {
	return (min_value <= val) && (val <= max_value);
    }
};

struct print_float4 : public thrust::unary_function<float4, void>
{
	__host__ __device__
	void operator() (float4 p) {
	    std::cout << "(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
	}
};

int main()
{
    sfield<int, float> scalar_field(GRID_SIZE, GRID_SIZE, GRID_SIZE);
    thrust::copy(scalar_field.point_data_begin(), scalar_field.point_data_end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    threshold_geometry<sfield<int, float>, threshold_between> threshold(scalar_field, threshold_between(0, 1));
    threshold();

    thrust::for_each(threshold.verticesBegin(), threshold.verticesEnd(), print_float4());

    return 0;
}
