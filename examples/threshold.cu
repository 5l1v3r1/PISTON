#include "hip/hip_runtime.h"
/*
 * threshold.cu
 *
 *  Created on: Sep 21, 2011
 *      Author: ollie
 */

#include <GL/glut.h>
#include <piston/sphere.h>
#include <piston/threshold_geometry.h>

static const int GRID_SIZE = 16;

template <typename IndexType, typename ValueType>
struct height_field : public piston::image3d<IndexType, ValueType, thrust::host_space_tag>
{
    struct height_functor : public piston::implicit_function3d<IndexType, ValueType> {
	typedef piston::implicit_function3d<IndexType, ValueType> Parent;
	typedef typename Parent::InputType InputType;

	__host__ __device__
	ValueType operator()(InputType pos) const {
	    return thrust::get<2>(pos);
	};
    };

    typedef piston::image3d<IndexType, ValueType, thrust::host_space_tag> Parent;

    typedef thrust::transform_iterator<height_functor,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    height_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     height_functor()){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter + this->NPoints;
    }
};

template <typename IndexType, typename ValueType>
struct sfield : public piston::image3d<IndexType, ValueType, thrust::host_space_tag>
{
    typedef piston::image3d<IndexType, ValueType, thrust::host_space_tag> Parent;

    typedef thrust::transform_iterator<piston::sphere<IndexType, ValueType>,
				       typename Parent::GridCoordinatesIterator> PointDataIterator;
    PointDataIterator iter;

    sfield(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	iter(this->grid_coordinates_iterator,
	     piston::sphere<IndexType, ValueType>(xdim/2, ydim/2, zdim/2, 1)){}

    PointDataIterator point_data_begin() {
	return iter;
    }

    PointDataIterator point_data_end() {
	return iter+this->NPoints;
    }
};

struct threshold_between : thrust::unary_function<float, bool>
{
    float min_value;
    float max_value;

    threshold_between(float min_value, float max_value) :
	min_value(min_value), max_value(max_value) {}

    __host__ __device__
    bool operator() (float val) const {
	return (min_value <= val) && (val <= max_value);
    }
};

struct print_float4 : public thrust::unary_function<float4, void>
{
	__host__ __device__
	void operator() (float4 p) {
	    std::cout << "(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
	}
};


int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0, 0, 0.0);
float3 translate = make_float3(0.0, 0.0, 0.0);

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
	mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
	mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
	rotate.x += dy * 0.2;
	rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
	translate.x += dx * 0.01;
	translate.y -= dy * 0.01;
    } else if (mouse_buttons==4) {
	translate.z += dy * 0.1;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

bool wireframe = false;
bool animate = true;
void keyboard( unsigned char key, int x, int y )
{
    switch (key) {
    case 'w':
	wireframe = !wireframe;
	break;
    case 'a':
	animate = !animate;
	break;
    }
}


threshold_geometry<sfield<int, float>, threshold_between> *threshold_p;

void display()
{
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) {
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    } else {
	glPolygonMode(GL_BACK, GL_FILL);
	glPolygonMode(GL_FRONT, GL_FILL);
    }

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    glPushMatrix();

    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);
    glTranslatef(-(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2);
    glTranslatef(translate.x, translate.y, translate.z);

    thrust::host_vector<float4> vertices(threshold_p->verticesBegin(),
                                         threshold_p->verticesEnd());

//    thrust::for_each(vertices.begin(), vertices.end(), print_float4());

    glColor4f(1.0f, 1.0f, 1.0f, 1.0f);

//    glNormalPointer(GL_FLOAT, 0, &normals[0]);
//    glColorPointer(3, GL_FLOAT, 0, &normals[0]);
    glVertexPointer(4, GL_FLOAT, 0, &vertices[0]);
    glDrawArrays(GL_QUADS, 0, vertices.size());

    // set view matrix for 2D message
    // TBD
    glutSwapBuffers();
}

void idle()
{
    if (animate) {
//	isovalue += delta;
//	if (isovalue > maxiso)
//	    delta = -0.05;
//	if (isovalue < miniso)
//	    delta = 0.05;
//	glutPostRedisplay();
    }
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Marching Cube");

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float ambient[] = { 0.5, 0.0, 0.0, 1.0 };
    float diffuse[] = { 0.5, 0.0, 0.0, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

//    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, ambient);
//    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse);
//    glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
//    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, ambient);
//    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
//    glEnable(GL_NORMALIZE);
    glEnable(GL_COLOR_MATERIAL);

    /* Setup the view of the cube. */
    glMatrixMode(GL_PROJECTION);
    gluPerspective( /* field of view in degree */ 60.0,
                    /* aspect ratio */ 1.0,
                    /* Z near */ 1.0, /* Z far */ GRID_SIZE*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, GRID_SIZE*1.5,  /* eye is at (0,0, 1.5*GRID_SIZE) */
              0.0, 0.0, 0.0,		/* center is at (0,0,0) */
              0.0, 1.0, 0.0);		/* up is in positive Y direction */
    glPushMatrix();

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
//    glEnableClientState(GL_NORMAL_ARRAY);
//    glEnableClientState(GL_COLOR_ARRAY);

//    glutReshapeFunc( reshape);
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutIdleFunc(idle);
    glutMainLoop();
}

int main(int argc, char *argv[])
{
    sfield<int, float> scalar_field(GRID_SIZE, GRID_SIZE, GRID_SIZE);
    thrust::copy(scalar_field.point_data_begin(), scalar_field.point_data_end(), std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    threshold_geometry<sfield<int, float>, threshold_between> threshold(scalar_field, threshold_between(9, 25));
    threshold();

    threshold_p = &threshold;

    initGL(argc, argv);

    return 0;
}
