#include "hip/hip_runtime.h"
/*
 * plane_field_adaptor.cu
 *
 *  Created on: Nov 15, 2012
 *      Author: ollie
 */
#include <piston/piston_math.h>
#include <piston/util/tangle_field.h>
#include <piston/plane_filed_adaptor.h>

#define SPACE thrust::host_space_tag
using namespace piston;

struct print_tuple
{
    template <typename Tuple>
    __host__ __device__
    void operator ()(Tuple xyz) {
	std::cout << "("
		  << thrust::get<0>(xyz) << ", "
		  << thrust::get<1>(xyz) << ", "
		  << thrust::get<2>(xyz) << ")" << std::endl;
    }
};

struct print_float
{
    __host__ __device__
    void operator()(float x) {
	std::cout << x << " ";
    }
};
int
main()
{
    tangle_field<SPACE> tangle(4,4,4);

    plane_field_adaptor<tangle_field<SPACE> > plane(tangle,
                                                    make_float3(0, 0, 0),
                                                    make_float3(0, 0, 1));

    thrust::for_each(tangle.physical_coordinates_begin(),
                     tangle.physical_coordinates_end(), print_tuple());

    thrust::for_each(plane.physical_coordinates_begin(),
                     plane.physical_coordinates_end(), print_tuple());

    thrust::copy(tangle.point_data_begin(),
                 tangle.point_data_end(),
                 std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    thrust::copy(plane.point_data_begin(),
                 plane.point_data_end(),
                 std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;

    return 0;
}



