#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/glut.h>

#include <cuda_gl_interop.h>

#include <cutil_math.h>
#include <piston/choose_container.h>

#define SPACE thrust::detail::default_device_space_tag
using namespace piston;

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float4>
{
    const ValueType min;
    const ValueType max;

    __host__ __device__
    color_map(ValueType min, ValueType max, bool reversed=false) :
	min(min), max(max) {}

    __host__ __device__
    float4 operator()(ValueType val) {
	// HSV rainbow for height field, stolen form Manta
	const float V = 0.7f, S = 1.0f;
	float H = (1.0f - static_cast<float> (val) / (max - min));

	if (H < 0.0f)
	    H = 0.0f;
	else if (H > 1.0f)
	    H = 1.0f;
	H *= 4.0f;

	float i = floor(H);
	float f = H - i;

	float p = V * (1.0 - S);
	float q = V * (1.0 - S * f);
	float t = V * (1.0 - S * (1 - f));

	float R, G, B;
	if (i == 0.0) {
	    R = V;
	    G = t;
	    B = p;
	} else if (i == 1.0) {
	    R = q;
	    G = V;
	    B = p;
	} else if (i == 2.0) {
	    R = p;
	    G = V;
	    B = t;
	} else if (i == 3.0) {
	    R = p;
	    G = q;
	    B = V;
	} else if (i == 4.0) {
	    R = t;
	    G = p;
	    B = V;
	} else {
	    // i == 5.0
	    R = V;
	    G = p;
	    B = q;
	}
	return make_float4(R, G, B, 1.0);
    }
};

#include <piston/implicit_function.h>
#include <piston/image3d.h>
#include <piston/marching_cube.h>
#include <sys/time.h>
#include <stdio.h>

static const int GRID_SIZE = 256;

template <typename IndexType, typename ValueType, typename Space>
struct tangle_field : public piston::image3d<IndexType, ValueType, Space>
{
    typedef piston::image3d<IndexType, ValueType, Space> Parent;

//    typedef typename detail::choose_container<typename Parent::CountingIterator, thrust::tuple<IndexType, IndexType, IndexType> >::type GridCoordinatesContainer;
//    GridCoordinatesContainer grid_coordinates_vector;
//    typedef typename GridCoordinatesContainer::iterator GridCoordinatesIterator;

    typedef typename detail::choose_container<typename Parent::CountingIterator, ValueType>::type PointDataContainer;
    PointDataContainer point_data_vector;
    typedef typename PointDataContainer::iterator PointDataIterator;

    struct tangle_functor : public piston::implicit_function3d<IndexType, ValueType>
    {
	typedef piston::implicit_function3d<IndexType, ValueType> Parent;
	typedef typename Parent::InputType InputType;

        const float xscale;
        const float yscale;
        const float zscale;

        tangle_functor(IndexType xdim, IndexType ydim, IndexType zdim) :
            xscale(2.0f/(xdim - 1.0f)),
            yscale(2.0f/(ydim - 1.0f)),
            zscale(2.0f/(zdim - 1.0f)) {}

        __host__ __device__
        float operator()(InputType pos) const {
            // scale and shift such that x, y, z <- [-1,1]
            const float x = 3.0f*(thrust::get<0>(pos)*xscale - 1.0f);
            const float y = 3.0f*(thrust::get<1>(pos)*yscale - 1.0f);
            const float z = 3.0f*(thrust::get<2>(pos)*zscale - 1.0f);

            const float v = (x*x*x*x - 5.0f*x*x +y*y*y*y - 5.0f*y*y +z*z*z*z - 5.0f*z*z + 11.8f) * 0.2f + 0.5f;

            return v;
        }
    };

    tangle_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
//	grid_coordinates_vector(Parent::grid_coordinates_begin(), Parent::grid_coordinates_end()),
//	point_data_vector(thrust::make_transform_iterator(grid_coordinates_vector.begin(), tangle_functor(xdim, ydim, zdim)),
//	                  thrust::make_transform_iterator(grid_coordinates_vector.end(),   tangle_functor(xdim, ydim, zdim)))
//	grid_coordinates_vector(Parent::grid_coordinates_begin(), Parent::grid_coordinates_end()),
	point_data_vector(thrust::make_transform_iterator(Parent::grid_coordinates_begin(), tangle_functor(xdim, ydim, zdim)),
	                  thrust::make_transform_iterator(Parent::grid_coordinates_end(),   tangle_functor(xdim, ydim, zdim)))
	                  {}

    void resize(int xdim, int ydim, int zdim) {
	Parent::resize(xdim, ydim, zdim);
	point_data_vector.resize(this->NPoints);
//	point_data_vector.assign(thrust::make_transform_iterator(grid_coordinates_vector.begin(), tangle_functor(xdim, ydim, zdim)),
//	                         thrust::make_transform_iterator(grid_coordinates_vector.end(),   tangle_functor(zdim, ydim, zdim)));
    }

//    GridCoordinatesIterator grid_coordinates_begin() {
//	return grid_coordinates_vector.begin();
//    }
//    GridCoordinatesIterator grid_coordinates_end() {
//	return grid_coordinates_vector.end();
//    }

    PointDataIterator point_data_begin() {
	return point_data_vector.begin();
    }
    PointDataIterator point_data_end() {
	return point_data_vector.end();
    }
};



int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0, 0, 0.0);
float3 translate = make_float3(0.0, 0.0, 0.0);

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
	mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
	mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
	rotate.x += dy * 0.2;
	rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
	translate.x += dx * 0.01;
	translate.y -= dy * 0.01;
    } else if (mouse_buttons==4) {
	translate.z += dy * 0.1;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

bool wireframe = false;
bool animate = true;
void keyboard( unsigned char key, int x, int y )
{
    switch (key) {
    case 'w':
	wireframe = !wireframe;
	break;
    case 'a':
	animate = !animate;
	break;
    }
}

struct tuple2float4 : thrust::unary_function<thrust::tuple<int, int, int>, float4>
{
	__host__ __device__
	float4 operator()(thrust::tuple<int, int, int> xyz) {
	    return make_float4((float) thrust::get<0>(xyz),
	                       (float) thrust::get<1>(xyz),
	                       (float) thrust::get<2>(xyz),
	                       1.0f);
	}
};


marching_cube<tangle_field<int, float, SPACE>, tangle_field<int, float, SPACE> > *isosurface_p;

GLuint quads_vbo[3];
struct hipGraphicsResource *quads_pos_res, *quads_normal_res, *quads_color_res;
unsigned int buffer_size;

void create_vbo()
{
    glGenBuffers(3, quads_vbo);
    int error;

    std::cout << "number of vertices: " << thrust::distance(isosurface_p->vertices_begin(), isosurface_p->vertices_end()) << std::endl;
    buffer_size = thrust::distance(isosurface_p->vertices_begin(), isosurface_p->vertices_end())* sizeof(float4);

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_pos_res, quads_vbo[0],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register pos buffer cuda error: " << error << "\n";
    }

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_normal_res, quads_vbo[1],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register normal buffer cuda error: " << error << "\n";
    }

    // initialize color buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_color_res, quads_vbo[2],
                                     cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register color buffer cuda error: " << error << "\n";
    }
}

int frame_count = 0;
float seconds = 0.0f;
bool new_frame = false;

void display()
{
    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);

    (*isosurface_p)();
#if 1
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) {
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    } else {
//	glPolygonMode(GL_BACK, GL_LINE);
//	glPolygonMode(GL_FRONT, GL_FILL);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    }

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    glPushMatrix();

    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);
    glTranslatef(-(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2);
    glTranslatef(translate.x, translate.y, translate.z);

    float4 *raw_ptr;
    size_t num_bytes;

    if (new_frame) {
	hipGraphicsMapResources(1, &quads_pos_res, 0);
	hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_pos_res);
	//    thrust::copy(thrust::make_transform_iterator(isosurface_p->vertices_begin(), tuple2float4()),
	//                 thrust::make_transform_iterator(isosurface_p->vertices_end(),   tuple2float4()),
	//                 thrust::device_ptr<float4>(raw_ptr));
	thrust::copy(isosurface_p->vertices_begin(),
	             isosurface_p->vertices_end(),
	             thrust::device_ptr<float4>(raw_ptr));
	hipGraphicsUnmapResources(1, &quads_pos_res, 0);
    }
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    float3 *normal;
    if (new_frame) {
	hipGraphicsMapResources(1, &quads_normal_res, 0);
	hipGraphicsResourceGetMappedPointer((void**)&normal, &num_bytes, quads_normal_res);
	thrust::copy(isosurface_p->normals_begin(),
	             isosurface_p->normals_end(),
	             thrust::device_ptr<float3>(normal));
	hipGraphicsUnmapResources(1, &quads_normal_res, 0);
    }
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glNormalPointer(GL_FLOAT, 0, 0);

    if (new_frame) {
	hipGraphicsMapResources(1, &quads_color_res, 0);
	hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_color_res);
	thrust::transform(isosurface_p->scalars_begin(), isosurface_p->scalars_end(),
	                  thrust::device_ptr<float4>(raw_ptr),
	                  color_map<float>(0.0f, .9f));
	hipGraphicsUnmapResources(1, &quads_color_res, 0);
    }
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glColorPointer(4, GL_FLOAT, 0, 0);

    glDrawArrays(GL_TRIANGLES, 0, buffer_size/sizeof(float4));

    glutSwapBuffers();
    glBindBuffer(GL_ARRAY_BUFFER, 0);

    new_frame = false;
#endif
    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frame_count++;
    seconds += diff.tv_sec + 1.0E-6*diff.tv_usec;

    if (frame_count > 10) {
	char title[256];
	sprintf(title, "Tangle, fps: %2.2f", 10.0f/seconds);
	glutSetWindowTitle(title);
	seconds = 0.0f;
	frame_count = 0;
    }

}

void idle()
{
    if (animate) {
//	isovalue += delta;
//	if (isovalue > maxiso)
//	    delta = -0.05;
//	if (isovalue < miniso)
//	    delta = 0.05;
    }
    glutPostRedisplay();
}

void reshape(int w, int h)
{
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (float) w / (float) h, 0.1, GRID_SIZE*4.0f);

    glMatrixMode(GL_MODELVIEW);
    glViewport(0, 0, w, h);
}

void timer(int value)
{
    glutPostRedisplay();
    glutTimerFunc(10, timer, 0);
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Tangle");

    glewInit();

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float black[] = { 0.0, 0.0, 0.0, 1.0 };
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float ambient[] = { 0.5, 0.0, 0.0, 1.0 };
    float diffuse[] = { 0.5, 0.0, 0.0, 1.0 };
    float lightPos[] = { 0.0f, 0.0f, 1.0f, 0.0f };

    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, ambient);
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse);
    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, black);

    glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
//    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
//    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModelfv(GL_LIGHT_MODEL_AMBIENT, black);
//    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
//    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

    /* Setup the view of the cube. */
    glMatrixMode(GL_PROJECTION);
    gluPerspective( /* field of view in degree */ 60.0,
                    /* aspect ratio */ 1.0,
                    /* Z near */ 1.0, /* Z far */ GRID_SIZE*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, GRID_SIZE*1.5,  /* eye is at (0,0, 1.5*GRID_SIZE) */
              0.0, 0.0, 0.0,		/* center is at (0,0,0) */
              0.0, 1.0, 0.0);		/* up is in positive Y direction */
    glPushMatrix();

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}

int main(int argc, char *argv[])
{
    initGL(argc, argv);
    cudaGLSetGLDevice(0);

    tangle_field<int, float, SPACE> tangle(GRID_SIZE, GRID_SIZE, GRID_SIZE);

    marching_cube<tangle_field<int, float, SPACE>,  tangle_field<int, float, SPACE> > isosurface(tangle, tangle, 0.2f);

    isosurface();
    isosurface_p = &isosurface;
    new_frame = true;

    create_vbo();

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutIdleFunc(idle);
    glutReshapeFunc(reshape);

    glutMainLoop();

    return 0;
}
