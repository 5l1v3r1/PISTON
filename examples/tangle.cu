#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.
Copyright 2011. Los Alamos National Security, LLC. This software was produced under U.S. Government contract DE-AC52-06NA25396 for Los Alamos National Laboratory (LANL),
which is operated by Los Alamos National Security, LLC for the U.S. Department of Energy. The U.S. Government has rights to use, reproduce, and distribute this software.

NEITHER THE GOVERNMENT NOR LOS ALAMOS NATIONAL SECURITY, LLC MAKES ANY WARRANTY, EXPRESS OR IMPLIED, OR ASSUMES ANY LIABILITY FOR THE USE OF THIS SOFTWARE.

If software is modified to produce derivative works, such modified software should be clearly marked, so as not to confuse it with the version available from LANL.

Additionally, redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:
·         Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
·         Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other
          materials provided with the distribution.
·         Neither the name of Los Alamos National Security, LLC, Los Alamos National Laboratory, LANL, the U.S. Government, nor the names of its contributors may be used
          to endorse or promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY LOS ALAMOS NATIONAL SECURITY, LLC AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL LOS ALAMOS NATIONAL SECURITY, LLC OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA,
OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifdef __APPLE__
    #include <GL/glew.h>
    #include <OpenGL/OpenGL.h>
    #include <GLUT/glut.h>
#else
    #include <GL/glew.h>
    #include <GL/glut.h>
    #include <GL/gl.h>
#endif

#include <QtGui>
#include <QtOpenGL>
#include <QObject>

#include <cuda_gl_interop.h>

#include <vtkXMLImageDataReader.h>

#include <cutil_math.h>
#include <piston/choose_container.h>

#define SPACE thrust::detail::default_device_space_tag
using namespace piston;

#include <piston/implicit_function.h>
#include <piston/image3d.h>
#include <piston/vtk_image3d.h>
#include <piston/marching_cube.h>

#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#include "glwindow.h"


template <typename IndexType, typename ValueType, typename Space>
struct tangle_field : public piston::image3d<IndexType, ValueType, Space>
{
    typedef piston::image3d<IndexType, ValueType, Space> Parent;

    typedef typename detail::choose_container<typename Parent::CountingIterator, ValueType>::type PointDataContainer;
    PointDataContainer point_data_vector;
    typedef typename PointDataContainer::iterator PointDataIterator;

    struct tangle_functor : public piston::implicit_function3d<IndexType, ValueType>
    {
	typedef piston::implicit_function3d<IndexType, ValueType> Parent;
	typedef typename Parent::InputType InputType;

        const float xscale;
        const float yscale;
        const float zscale;

        tangle_functor(IndexType xdim, IndexType ydim, IndexType zdim) :
            xscale(2.0f/(xdim - 1.0f)),
            yscale(2.0f/(ydim - 1.0f)),
            zscale(2.0f/(zdim - 1.0f)) {}

        __host__ __device__
        float operator()(InputType pos) const {
            // scale and shift such that x, y, z <- [-1,1]
            const float x = 3.0f*(thrust::get<0>(pos)*xscale - 1.0f);
            const float y = 3.0f*(thrust::get<1>(pos)*yscale - 1.0f);
            const float z = 3.0f*(thrust::get<2>(pos)*zscale - 1.0f);

            const float v = (x*x*x*x - 5.0f*x*x +y*y*y*y - 5.0f*y*y +z*z*z*z - 5.0f*z*z + 11.8f) * 0.2f + 0.5f;

            return v;
        }
    };

    tangle_field(int xdim, int ydim, int zdim) :
	Parent(xdim, ydim, zdim),
	point_data_vector(thrust::make_transform_iterator(Parent::grid_coordinates_begin(), tangle_functor(xdim, ydim, zdim)),
	                  thrust::make_transform_iterator(Parent::grid_coordinates_end(),   tangle_functor(xdim, ydim, zdim)))
	                  {}

    void resize(int xdim, int ydim, int zdim) {
	Parent::resize(xdim, ydim, zdim);

	point_data_vector.resize(this->NPoints);
	point_data_vector.assign(thrust::make_transform_iterator(Parent::grid_coordinates.begin(), tangle_functor(xdim, ydim, zdim)),
	                         thrust::make_transform_iterator(Parent::grid_coordinates.end(),   tangle_functor(zdim, ydim, zdim)));
    }

    PointDataIterator point_data_begin() {
	return point_data_vector.begin();
    }
    PointDataIterator point_data_end() {
	return point_data_vector.end();
    }
};


struct timeval begin, end, diff;
int frame_count = 0;
int grid_size = 256;
float cameraFOV = 60.0;
bool wireframe = false;

tangle_field<int, float, SPACE>* tangle;
marching_cube<tangle_field<int, float, SPACE>, tangle_field<int, float, SPACE> > *isosurface;

GLuint quads_vbo[3];
struct hipGraphicsResource *quads_pos_res, *quads_normal_res, *quads_color_res;
unsigned int buffer_size;


void create_vbo()
{
    glGenBuffers(3, quads_vbo);
    int error;

    //std::cout << "number of vertices: " << thrust::distance(isosurface_p->vertices_begin(), isosurface_p->vertices_end()) << std::endl;
    buffer_size = thrust::distance(isosurface->vertices_begin(), isosurface->vertices_end())* sizeof(float4);

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_pos_res, quads_vbo[0],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register pos buffer cuda error: " << error << "\n";
    }

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_normal_res, quads_vbo[1],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register normal buffer cuda error: " << error << "\n";
    }

    // initialize color buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_color_res, quads_vbo[2],
                                     cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register color buffer cuda error: " << error << "\n";
    }
}


GLWindow::GLWindow(QWidget *parent)
    : QGLWidget(QGLFormat(QGL::SampleBuffers), parent)
{
    setFocusPolicy(Qt::StrongFocus);
    timer = new QTimer(this);
    connect(timer, SIGNAL(timeout()), this, SLOT(updateGL()));
    timer->start(1);
}


GLWindow::~GLWindow()
{

}


QSize GLWindow::minimumSizeHint() const
{
    return QSize(100, 100);
}


QSize GLWindow::sizeHint() const
{
    return QSize(1024, 1024);
}


bool GLWindow::initialize(int argc, char *argv[])
{
    return true;
}


void GLWindow::initializeGL()
{
    glewInit();
    cudaGLSetGLDevice(0);

    tangle = new tangle_field<int, float, SPACE>(grid_size, grid_size, grid_size);
    isosurface = new marching_cube<tangle_field<int, float, SPACE>,  tangle_field<int, float, SPACE> >(*tangle, *tangle, 0.2f);

    (*isosurface)();

    create_vbo();

    qrot.set(0,0,0,1);

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

    // Setup the view of the cube.
    glMatrixMode(GL_PROJECTION);
    gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, grid_size*1.5,
              0.0, 0.0, 0.0,
              0.0, 1.0, 0.0);

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}


void GLWindow::paintGL()
{
    timer->stop();

    if (frame_count == 0) gettimeofday(&begin, 0);

    (*isosurface)();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    else glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective( cameraFOV, 1.0, 1.0, grid_size*4.0);

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();

    qrot.getRotMat(rotationMatrix);
    glMultMatrixf(rotationMatrix);

    glTranslatef(-(grid_size-1)/2, -(grid_size-1)/2, -(grid_size-1)/2);

    float4 *raw_ptr;
    size_t num_bytes;

    hipGraphicsMapResources(1, &quads_pos_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_pos_res);

    thrust::copy(isosurface->vertices_begin(),
                 isosurface->vertices_end(),
                 thrust::device_ptr<float4>(raw_ptr));

    hipGraphicsUnmapResources(1, &quads_pos_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    float3 *normal;
    hipGraphicsMapResources(1, &quads_normal_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&normal, &num_bytes, quads_normal_res);
    thrust::copy(isosurface->normals_begin(),
                 isosurface->normals_end(),
                 thrust::device_ptr<float3>(normal));
    hipGraphicsUnmapResources(1, &quads_normal_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glNormalPointer(GL_FLOAT, 0, 0);

    hipGraphicsMapResources(1, &quads_color_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_color_res);
    thrust::transform(isosurface->scalars_begin(), isosurface->scalars_end(),
                      thrust::device_ptr<float4>(raw_ptr),
                      color_map<float>(31.0f, 500.0f));
    hipGraphicsUnmapResources(1, &quads_color_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glColorPointer(4, GL_FLOAT, 0, 0);

    glDrawArrays(GL_TRIANGLES, 0, buffer_size/sizeof(float4));

    glPopMatrix();

    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frame_count++;
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    if (seconds > 0.5f)
    {
      char title[256];
      sprintf(title, "Marching Cube, fps: %2.2f", float(frame_count)/seconds);
      std::cout << title << std::endl;
      seconds = 0.0f;
      frame_count = 0;
    }

    timer->start(1);
}


void GLWindow::resizeGL(int width, int height)
{
    glViewport(0, 0, width, height);
}


void GLWindow::mousePressEvent(QMouseEvent *event)
{
    lastPos = event->pos();
}


void GLWindow::mouseMoveEvent(QMouseEvent *event)
{
    int dx = event->x() - lastPos.x();
    int dy = event->y() - lastPos.y();

    if (event->buttons() & Qt::LeftButton)
    {
      Quaternion newRotX;
      newRotX.setEulerAngles(-0.2*dx*3.14159/180.0, 0.0, 0.0);
      qrot.mul(newRotX);

      Quaternion newRotY;
      newRotY.setEulerAngles(0.0, 0.0, -0.2*dy*3.14159/180.0);
      qrot.mul(newRotY);
    }
    else if (event->buttons() & Qt::RightButton)
    {
      cameraFOV += dy/20.0;
    }
    lastPos = event->pos();
}


void GLWindow::keyPressEvent(QKeyEvent *event)
{
   if ((event->key() == 'w') || (event->key() == 'W'))
       wireframe = !wireframe;
}


