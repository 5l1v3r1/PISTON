#include "hip/hip_runtime.h"
/*
 * eddy.cu
 *
 *  Created on: Oct 21, 2011
 *      Author: ollie
 */

#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/glut.h>

#include <cuda_gl_interop.h>
#include <sys/time.h>

#include <vtkXMLImageDataReader.h>

#include <piston/threshold_geometry.h>
#include <piston/vtk_image3d.h>

static const int GRID_SIZE = 256;

using namespace piston;

#define SPACE thrust::detail::default_device_space_tag
//#define SPACE thrust::host_space


int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0, 0, 0.0);
float3 translate = make_float3(0.0, 0.0, 0.0);

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
	mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
	mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
	rotate.x += dy * 0.2;
	rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
	translate.x += dx * 0.01;
	translate.y -= dy * 0.01;
    } else if (mouse_buttons==4) {
	translate.z += dy * 0.1;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

bool wireframe = false;
bool animate = true;
void keyboard( unsigned char key, int x, int y )
{
    switch (key) {
    case 'w':
	wireframe = !wireframe;
	break;
    case 'a':
	animate = !animate;
	break;
    }
}

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float4>
{
    const ValueType min;
    const ValueType max;

    __host__ __device__
    color_map(ValueType min, ValueType max) :
	min(min), max(max) {}

    __host__ __device__
    float4 operator()(ValueType val) {
	// HSV rainbow for height field, stolen form Manta
	const float V = 0.7f, S = 1.0f;
	float H = (1.0f - static_cast<float> (val) / (max - min));

	if (H < 0.0f)
	    H = 0.0f;
	else if (H > 1.0f)
	    H = 1.0f;
	H *= 4.0f;

	float i = floor(H);
	float f = H - i;

	float p = V * (1.0 - S);
	float q = V * (1.0 - S * f);
	float t = V * (1.0 - S * (1 - f));

	float R, G, B;
	if (i == 0.0) {
	    R = V;
	    G = t;
	    B = p;
	} else if (i == 1.0) {
	    R = q;
	    G = V;
	    B = p;
	} else if (i == 2.0) {
	    R = p;
	    G = V;
	    B = t;
	} else if (i == 3.0) {
	    R = p;
	    G = q;
	    B = V;
	} else if (i == 4.0) {
	    R = t;
	    G = p;
	    B = V;
	} else {
	    // i == 5.0
	    R = V;
	    G = p;
	    B = q;
	}
	return make_float4(R, G, B, 1.0);
    }
};

struct tuple2float4 : thrust::unary_function<thrust::tuple<int, int, int>, float4>
{
	__host__ __device__
	float4 operator()(thrust::tuple<int, int, int> xyz) {
	    return make_float4((float) thrust::get<0>(xyz),
	                       (float) thrust::get<1>(xyz),
	                       (float) thrust::get<2>(xyz),
	                       1.0f);
	}
};


threshold_geometry<vtk_image3d<int, float, SPACE> > *threshold_p;
GLuint quads_vbo[2];
struct hipGraphicsResource *quads_pos_res, *quads_color_res;
unsigned int buffer_size;

void create_vbo()
{
    glGenBuffers(2, quads_vbo);

    int error;
    std::cout << "number of vertices: " << thrust::distance(threshold_p->vertices_begin(), threshold_p->vertices_end()) << std::endl;
    buffer_size = thrust::distance(threshold_p->vertices_begin(), threshold_p->vertices_end())* sizeof(float4);

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_pos_res, quads_vbo[0],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register pos buffer cuda error: " << error << "\n";
    }

    // initialize color buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if (hipGraphicsGLRegisterBuffer(&quads_color_res, quads_vbo[1],
                                     cudaGraphicsMapFlagsWriteDiscard) != hipSuccess) {
	std::cout << "register color buffer cuda error: " << error << "\n";
    }
}

struct timeval begin, end, diff;
int frame_count = 0;

void display()
{
    if (frame_count == 0) {
	gettimeofday(&begin, 0);
    }

//    (*threshold_p)();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) {
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    } else {
	glPolygonMode(GL_BACK, GL_FILL);
	glPolygonMode(GL_FRONT, GL_FILL);
    }

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    glPushMatrix();

    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);
    glTranslatef(-(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2);
    glTranslatef(translate.x, translate.y, translate.z);

    float4 *raw_ptr;
    size_t num_bytes;

    hipGraphicsMapResources(1, &quads_pos_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_pos_res);
    thrust::copy(thrust::make_transform_iterator(threshold_p->vertices_begin(), tuple2float4()),
                 thrust::make_transform_iterator(threshold_p->vertices_end(),   tuple2float4()),
                 thrust::device_ptr<float4>(raw_ptr));
    hipGraphicsUnmapResources(1, &quads_pos_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    hipGraphicsMapResources(1, &quads_color_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_color_res);
    thrust::transform(threshold_p->scalars_begin(), threshold_p->scalars_end(),
                      thrust::device_ptr<float4>(raw_ptr),
                      color_map<float>(-500.0f, -0.1f));
    hipGraphicsUnmapResources(1, &quads_color_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glColorPointer(4, GL_FLOAT, 0, 0);


    glDrawArrays(GL_QUADS, 0, buffer_size/sizeof(float4));

    glutSwapBuffers();

    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frame_count++;
    float seconds = diff.tv_sec + 1.0E-6*diff.tv_usec;
    if (seconds > 0.1f) {
	char title[256];
	sprintf(title, "Eddy, fps: %2.2f", float(frame_count)/seconds);
	glutSetWindowTitle(title);
	seconds = 0.0f;
	frame_count = 0;
    }
}

void idle()
{
    if (animate) {
//	isovalue += delta;
//	if (isovalue > maxiso)
//	    delta = -0.05;
//	if (isovalue < miniso)
//	    delta = 0.05;
//	glutPostRedisplay();
    }
    glutPostRedisplay();
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Ocean Eddy");

    glewInit();

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
//    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

    /* Setup the view of the cube. */
    glMatrixMode(GL_PROJECTION);
    gluPerspective( /* field of view in degree */ 60.0,
                    /* aspect ratio */ 1.0,
                    /* Z near */ 1.0, /* Z far */ GRID_SIZE*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, GRID_SIZE*1.5,  /* eye is at (0,0, 1.5*GRID_SIZE) */
              0.0, 0.0, 0.0,		/* center is at (0,0,0) */
              0.0, 1.0, 0.0);		/* up is in positive Y direction */
    glPushMatrix();

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
//    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}

int main(int argc, char **argv)
{
    initGL(argc, argv);
    cudaGLSetGLDevice(0);

    vtkXMLImageDataReader *reader = vtkXMLImageDataReader::New();
    reader->SetFileName("/home/ollie/DataParallelData/NorthPacificSingle.vti");
    reader->Update();

    vtkImageData *vtk_image = reader->GetOutput();


    vtk_image3d<int, float, SPACE> image(vtk_image);
    threshold_geometry<vtk_image3d<int, float, SPACE> > threshold(image, -500.0f, -0.01f);
    threshold();
    threshold_p = &threshold;
    create_vbo();

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutIdleFunc(idle);
    glutMainLoop();

    return 0;
}
