#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Los Alamos National Security, LLC
All rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

    Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.
    Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation
    	and/or other materials provided with the distribution.
    Neither the name of the Los Alamos National Laboratory nor the names of its contributors may be used to endorse or promote products derived from this
    	software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS
OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/glut.h>

#include <cuda_gl_interop.h>

#include <vtkXMLImageDataReader.h>

#include <cutil_math.h>
#include <piston/choose_container.h>

#define SPACE thrust::detail::default_device_space_tag
using namespace piston;

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float4>
{
    const ValueType min;
    const ValueType max;

    __host__ __device__
    color_map(ValueType min, ValueType max, bool reversed=false) :
	min(min), max(max) {}

    __host__ __device__
    float4 operator()(ValueType val) {
	// HSV rainbow for height field, stolen form Manta
	const float V = 0.7f, S = 1.0f;
	float H = (1.0f - static_cast<float> (val) / (max - min));

	if (H < 0.0f)
	    H = 0.0f;
	else if (H > 1.0f)
	    H = 1.0f;
	H *= 4.0f;

	float i = floor(H);
	float f = H - i;

	float p = V * (1.0 - S);
	float q = V * (1.0 - S * f);
	float t = V * (1.0 - S * (1 - f));

	float R, G, B;
	if (i == 0.0) {
	    R = V;
	    G = t;
	    B = p;
	} else if (i == 1.0) {
	    R = q;
	    G = V;
	    B = p;
	} else if (i == 2.0) {
	    R = p;
	    G = V;
	    B = t;
	} else if (i == 3.0) {
	    R = p;
	    G = q;
	    B = V;
	} else if (i == 4.0) {
	    R = t;
	    G = p;
	    B = V;
	} else {
	    // i == 5.0
	    R = V;
	    G = p;
	    B = q;
	}
	return make_float4(R, G, B, 1.0);
    }
};

#include <piston/util/sphere_field.h>
#include <piston/vtk_image3d.h>
#include <piston/marching_cube.h>

#include <sys/time.h>
#include <stdio.h>

#define STRINGIZE(x) #x
#define STRINGIZE_VALUE_OF(x) STRINGIZE(x)

static const int GRID_SIZE = 256;

int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float3 rotate = make_float3(0, 0, 0.0);
float3 translate = make_float3(0.0, 0.0, 0.0);

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN) {
	mouse_buttons |= 1<<button;
    } else if (state == GLUT_UP) {
	mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

void motion(int x, int y)
{
    float dx = x - mouse_old_x;
    float dy = y - mouse_old_y;

    if (mouse_buttons==1) {
	rotate.x += dy * 0.2;
	rotate.y += dx * 0.2;
    } else if (mouse_buttons==2) {
	translate.x += dx * 0.01;
	translate.y -= dy * 0.01;
    } else if (mouse_buttons==4) {
	translate.z += dy * 0.1;
    }

    mouse_old_x = x;
    mouse_old_y = y;
    glutPostRedisplay();
}

bool wireframe = false;
bool animate = true;
void keyboard( unsigned char key, int x, int y )
{
    switch (key) {
    case 'w':
	wireframe = !wireframe;
	break;
    case 'a':
	animate = !animate;
	break;
    }
}

struct tuple2float4 : thrust::unary_function<thrust::tuple<int, int, int>, float4>
{
	__host__ __device__
	float4 operator()(thrust::tuple<int, int, int> xyz) {
	    return make_float4((float) thrust::get<0>(xyz),
	                       (float) thrust::get<1>(xyz),
	                       (float) thrust::get<2>(xyz),
	                       1.0f);
	}
};


marching_cube<vtk_image3d<int, float, SPACE>, vtk_image3d<int, float, SPACE> > *isosurface_p;

GLuint quads_vbo[3];
struct hipGraphicsResource *quads_pos_res, *quads_normal_res, *quads_color_res;
unsigned int buffer_size;

void create_vbo()
{
    glGenBuffers(3, quads_vbo);
    int error;

    std::cout << "number of vertices: " << thrust::distance(isosurface_p->vertices_begin(), isosurface_p->vertices_end()) << std::endl;
    buffer_size = thrust::distance(isosurface_p->vertices_begin(), isosurface_p->vertices_end())* sizeof(float4);

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_pos_res, quads_vbo[0],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register pos buffer cuda error: " << error << "\n";
    }

    // initialize vertex buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_normal_res, quads_vbo[1],
                                              cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register normal buffer cuda error: " << error << "\n";
    }

    // initialize color buffer object
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    // register this buffer object with CUDA
    if ((error = hipGraphicsGLRegisterBuffer(&quads_color_res, quads_vbo[2],
                                     cudaGraphicsMapFlagsWriteDiscard)) != hipSuccess) {
	std::cout << "register color buffer cuda error: " << error << "\n";
    }
}

int frame_count = 0;
float seconds = 0.0f;

void display()
{
    struct timeval begin, end, diff;
    gettimeofday(&begin, 0);

//    (*isosurface_p)();

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    if (wireframe) {
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
    } else {
//	glPolygonMode(GL_BACK, GL_LINE);
//	glPolygonMode(GL_FRONT, GL_FILL);
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    }

    // set view matrix for 3D scene
    glMatrixMode(GL_MODELVIEW);
    glPopMatrix();
    glPushMatrix();

    glRotatef(rotate.x, 1.0, 0.0, 0.0);
    glRotatef(rotate.y, 0.0, 1.0, 0.0);
    glTranslatef(-(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2, -(GRID_SIZE-1)/2);
    glTranslatef(translate.x, translate.y, translate.z);

    float4 *raw_ptr;
    size_t num_bytes;

    hipGraphicsMapResources(1, &quads_pos_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_pos_res);
//    thrust::copy(thrust::make_transform_iterator(isosurface_p->vertices_begin(), tuple2float4()),
//                 thrust::make_transform_iterator(isosurface_p->vertices_end(),   tuple2float4()),
//                 thrust::device_ptr<float4>(raw_ptr));
    thrust::copy(isosurface_p->vertices_begin(),
                 isosurface_p->vertices_end(),
                 thrust::device_ptr<float4>(raw_ptr));
    hipGraphicsUnmapResources(1, &quads_pos_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[0]);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    float3 *normal;
    hipGraphicsMapResources(1, &quads_normal_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&normal, &num_bytes, quads_normal_res);
    thrust::copy(isosurface_p->normals_begin(),
                 isosurface_p->normals_end(),
                 thrust::device_ptr<float3>(normal));
    hipGraphicsUnmapResources(1, &quads_normal_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[1]);
    glNormalPointer(GL_FLOAT, 0, 0);

    hipGraphicsMapResources(1, &quads_color_res, 0);
    hipGraphicsResourceGetMappedPointer((void**)&raw_ptr, &num_bytes, quads_color_res);
    thrust::transform(isosurface_p->scalars_begin(), isosurface_p->scalars_end(),
                      thrust::device_ptr<float4>(raw_ptr),
                      color_map<float>(4.0f, 1600.0f));
    hipGraphicsUnmapResources(1, &quads_color_res, 0);
    glBindBuffer(GL_ARRAY_BUFFER, quads_vbo[2]);
    glColorPointer(4, GL_FLOAT, 0, 0);

    glDrawArrays(GL_TRIANGLES, 0, buffer_size/sizeof(float4));

    glutSwapBuffers();

    gettimeofday(&end, 0);
    timersub(&end, &begin, &diff);
    frame_count++;
    seconds += diff.tv_sec + 1.0E-6*diff.tv_usec;

    if (frame_count > 10) {
	char title[256];
	sprintf(title, "Marching Cube, fps: %2.2f", 10.0f/seconds);
	glutSetWindowTitle(title);
	seconds = 0.0f;
	frame_count = 0;
    }

}

void idle()
{
    if (animate) {
//	isovalue += delta;
//	if (isovalue > maxiso)
//	    delta = -0.05;
//	if (isovalue < miniso)
//	    delta = 0.05;
//	glutPostRedisplay();
    }
    glutPostRedisplay();
}

void initGL(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    glutInitWindowSize(800, 800);
    glutCreateWindow("Threshold");

    glewInit();

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
    glEnable(GL_DEPTH_TEST);
    glShadeModel(GL_SMOOTH);

    // good old-fashioned fixed function lighting
    float white[] = { 0.8, 0.8, 0.8, 1.0 };
    float lightPos[] = { 100.0, 100.0, -100.0, 1.0 };

    glMaterialfv(GL_FRONT_AND_BACK, GL_SPECULAR, white);
    glMaterialf(GL_FRONT_AND_BACK, GL_SHININESS, 100);

    glLightfv(GL_LIGHT0, GL_AMBIENT, white);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, white);
    glLightfv(GL_LIGHT0, GL_SPECULAR, white);
    glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

//    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, 1);
    glLightModeli(GL_LIGHT_MODEL_TWO_SIDE, 1);

    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

    /* Setup the view of the cube. */
    glMatrixMode(GL_PROJECTION);
    gluPerspective( /* field of view in degree */ 60.0,
                    /* aspect ratio */ 1.0,
                    /* Z near */ 1.0, /* Z far */ GRID_SIZE*4.0);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0.0, 0.0, GRID_SIZE*1.5,  /* eye is at (0,0, 1.5*GRID_SIZE) */
              0.0, 0.0, 0.0,		/* center is at (0,0,0) */
              0.0, 1.0, 0.0);		/* up is in positive Y direction */
    glPushMatrix();

    // enable vertex and normal arrays
    glEnableClientState(GL_VERTEX_ARRAY);
    glEnableClientState(GL_NORMAL_ARRAY);
    glEnableClientState(GL_COLOR_ARRAY);
}

int main(int argc, char *argv[])
{

    cudaGLSetGLDevice(0);
    initGL(argc, argv);

    vtkXMLImageDataReader *reader = vtkXMLImageDataReader::New();
    char filename[1024];
    sprintf(filename, "%s/rti256.vti", STRINGIZE_VALUE_OF(DATA_DIRECTORY));
    reader->SetFileName(filename);
    reader->Update();

    vtkImageData *vtk_image = reader->GetOutput();

    vtk_image3d<int, float, SPACE> image(vtk_image);
    marching_cube<vtk_image3d<int, float, SPACE>, vtk_image3d<int, float, SPACE> > isosurface(image, image, 40);

    isosurface();
    isosurface_p = &isosurface;

    create_vbo();

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutIdleFunc(idle);
    glutMainLoop();

    return 0;
}
